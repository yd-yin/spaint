#include "hip/hip_runtime.h"
/**
 * spaint: VOPFeatureCalculator_CUDA.cpp
 */

#include "features/cuda/VOPFeatureCalculator_CUDA.h"

#include <ITMLib/Engine/DeviceAgnostic/ITMRepresentationAccess.h>

#include "features/shared/VOPFeatureCalculator_Shared.h"

#define DEBUGGING 1

namespace spaint {

//#################### CUDA KERNELS ####################

__global__ void ck_calculate_surface_normals(const Vector3s *voxelLocations, const int voxelLocationCount,
                                             const SpaintVoxel *voxelData, const ITMVoxelIndex::IndexData *indexData,
                                             Vector3f *surfaceNormals)
{
  int voxelLocationIndex = threadIdx.x + blockDim.x * blockIdx.x;
  if(voxelLocationIndex < voxelLocationCount)
  {
    write_surface_normal(voxelLocationIndex, voxelLocations, voxelData, indexData, surfaceNormals);
  }
}

__global__ void ck_convert_patches_to_lab(const int voxelLocationCount, const size_t featureCount, float *features)
{
  int voxelLocationIndex = threadIdx.x + blockDim.x * blockIdx.x;
  if(voxelLocationIndex < voxelLocationCount)
  {
    convert_patch_to_lab(voxelLocationIndex, featureCount, features);
  }
}

__global__ void ck_fill_in_normal_features(const int voxelLocationCount, const Vector3f *surfaceNormals, const size_t featureCount, float *features)
{
  int voxelLocationIndex = threadIdx.x + blockDim.x * blockIdx.x;
  if(voxelLocationIndex < voxelLocationCount)
  {
    fill_in_normal_feature(voxelLocationIndex, surfaceNormals, featureCount, features);
  }
}

__global__ void ck_generate_coordinate_systems(const Vector3f *surfaceNormals, const int voxelLocationCount, Vector3f *xAxes, Vector3f *yAxes)
{
  int voxelLocationIndex = threadIdx.x + blockDim.x * blockIdx.x;
  if(voxelLocationIndex < voxelLocationCount)
  {
    generate_coordinate_system(voxelLocationIndex, surfaceNormals, xAxes, yAxes);
  }
}

__global__ void ck_generate_rgb_patches(const Vector3s *voxelLocations, const int voxelLocationCount,
                                        const Vector3f *xAxes, const Vector3f *yAxes,
                                        const SpaintVoxel *voxelData, const ITMVoxelIndex::IndexData *indexData,
                                        size_t patchSize, float patchSpacing, size_t featureCount,
                                        float *features)
{
  int voxelLocationIndex = threadIdx.x + blockDim.x * blockIdx.x;
  if(voxelLocationIndex < voxelLocationCount)
  {
    generate_rgb_patch(voxelLocationIndex, voxelLocations, xAxes, yAxes, voxelData, indexData, patchSize, patchSpacing, featureCount, features);
  }
}

__global__ void ck_update_coordinate_systems(const int voxelLocationCount, const float *features, size_t featureCount, size_t patchSize, size_t binCount,
                                             Vector3f *xAxes, Vector3f *yAxes)
{
  // TODO: Comment on the fixed size of the intensities array.
  __shared__ float histogram[64];
  __shared__ float intensities[256];

  int tid = threadIdx.x + blockDim.x * blockIdx.x;
  int voxelLocationIndex = tid / (patchSize * patchSize);

  compute_intensities_for_patch(tid, features, featureCount, patchSize, intensities);
  __syncthreads();

  compute_histogram_for_patch(tid, patchSize, intensities, binCount, histogram);
  __syncthreads();

  update_patch_coordinate_system(tid, patchSize * patchSize, binCount, histogram, &xAxes[voxelLocationIndex], &yAxes[voxelLocationIndex]);
}

//#################### CONSTRUCTORS ####################

VOPFeatureCalculator_CUDA::VOPFeatureCalculator_CUDA(size_t maxVoxelLocationCount, size_t patchSize, float patchSpacing)
: VOPFeatureCalculator(maxVoxelLocationCount, patchSize, patchSpacing)
{}

//#################### PRIVATE MEMBER FUNCTIONS ####################

void VOPFeatureCalculator_CUDA::calculate_surface_normals(const ORUtils::MemoryBlock<Vector3s>& voxelLocationsMB,
                                                          const SpaintVoxel *voxelData, const ITMVoxelIndex::IndexData *indexData) const
{
  const int voxelLocationCount = static_cast<int>(voxelLocationsMB.dataSize);

  int threadsPerBlock = 256;
  int numBlocks = (voxelLocationCount + threadsPerBlock - 1) / threadsPerBlock;

  ck_calculate_surface_normals<<<numBlocks,threadsPerBlock>>>(
    voxelLocationsMB.GetData(MEMORYDEVICE_CUDA),
    voxelLocationCount,
    voxelData,
    indexData,
    m_surfaceNormalsMB.GetData(MEMORYDEVICE_CUDA)
  );

#if DEBUGGING
  m_surfaceNormalsMB.UpdateHostFromDevice();
#endif
}

void VOPFeatureCalculator_CUDA::convert_patches_to_lab(int voxelLocationCount, ORUtils::MemoryBlock<float>& featuresMB) const
{
  int threadsPerBlock = 256;
  int numBlocks = (voxelLocationCount + threadsPerBlock - 1) / threadsPerBlock;

  ck_convert_patches_to_lab<<<numBlocks,threadsPerBlock>>>(
    voxelLocationCount,
    get_feature_count(),
    featuresMB.GetData(MEMORYDEVICE_CUDA)
  );

#ifdef DEBUGGING
  featuresMB.UpdateHostFromDevice();
#endif
}

void VOPFeatureCalculator_CUDA::fill_in_normal_features(int voxelLocationCount, ORUtils::MemoryBlock<float>& featuresMB) const
{
  int threadsPerBlock = 256;
  int numBlocks = (voxelLocationCount + threadsPerBlock - 1) / threadsPerBlock;

  ck_fill_in_normal_features<<<numBlocks,threadsPerBlock>>>(
    voxelLocationCount,
    m_surfaceNormalsMB.GetData(MEMORYDEVICE_CUDA),
    get_feature_count(),
    featuresMB.GetData(MEMORYDEVICE_CUDA)
  );
}

void VOPFeatureCalculator_CUDA::generate_coordinate_systems(int voxelLocationCount) const
{
  int threadsPerBlock = 256;
  int numBlocks = (voxelLocationCount + threadsPerBlock - 1) / threadsPerBlock;

  ck_generate_coordinate_systems<<<numBlocks,threadsPerBlock>>>(
    m_surfaceNormalsMB.GetData(MEMORYDEVICE_CUDA),
    voxelLocationCount,
    m_xAxesMB.GetData(MEMORYDEVICE_CUDA),
    m_yAxesMB.GetData(MEMORYDEVICE_CUDA)
  );

#if DEBUGGING
  m_xAxesMB.UpdateHostFromDevice();
  m_yAxesMB.UpdateHostFromDevice();
#endif
}

void VOPFeatureCalculator_CUDA::generate_rgb_patches(const ORUtils::MemoryBlock<Vector3s>& voxelLocationsMB,
                                                     const SpaintVoxel *voxelData, const ITMVoxelIndex::IndexData *indexData,
                                                     ORUtils::MemoryBlock<float>& featuresMB) const
{
  const int voxelLocationCount = static_cast<int>(voxelLocationsMB.dataSize);

  int threadsPerBlock = 256;
  int numBlocks = (voxelLocationCount + threadsPerBlock - 1) / threadsPerBlock;

  ck_generate_rgb_patches<<<numBlocks,threadsPerBlock>>>(
    voxelLocationsMB.GetData(MEMORYDEVICE_CUDA),
    voxelLocationCount,
    m_xAxesMB.GetData(MEMORYDEVICE_CUDA),
    m_yAxesMB.GetData(MEMORYDEVICE_CUDA),
    voxelData,
    indexData,
    m_patchSize,
    m_patchSpacing,
    get_feature_count(),
    featuresMB.GetData(MEMORYDEVICE_CUDA)
  );

#if DEBUGGING
  featuresMB.UpdateHostFromDevice();
#endif
}

void VOPFeatureCalculator_CUDA::update_coordinate_systems(int voxelLocationCount, const ORUtils::MemoryBlock<float>& featuresMB) const
{
  int threadsPerBlock = m_patchSize * m_patchSize;
  int numBlocks = voxelLocationCount;

  // TEMPORARY
  const int binCount = 36;

  ck_update_coordinate_systems<<<numBlocks,threadsPerBlock>>>(
    voxelLocationCount,
    featuresMB.GetData(MEMORYDEVICE_CUDA),
    get_feature_count(),
    m_patchSize,
    binCount,
    m_xAxesMB.GetData(MEMORYDEVICE_CUDA),
    m_yAxesMB.GetData(MEMORYDEVICE_CUDA)
  );
}

}
