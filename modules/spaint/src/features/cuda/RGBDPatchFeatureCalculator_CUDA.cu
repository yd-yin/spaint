#include "hip/hip_runtime.h"
/**
 * spaint: RGBDPatchFeatureCalculator.cpp
 * Copyright (c) Torr Vision Group, University of Oxford, 2016. All rights reserved.
 */

#include "features/cuda/RGBDPatchFeatureCalculator_CUDA.h"
#include "features/shared/RGBDPatchFeatureCalculator_Shared.h"

namespace spaint
{
__global__ void ck_compute_colour_feature(RGBDPatchFeature *features,
    const Vector4u *rgb, const float *depth, const Vector4i *offsets_rgb,
    const uchar *channels_rgb, Vector2i img_size, bool normalize)
{
  const int x = threadIdx.x + blockIdx.x * blockDim.x;
  const int y = threadIdx.y + blockIdx.y * blockDim.y;

  if (x >= img_size.x || y >= img_size.y)
    return;

  compute_colour_patch_feature(features, rgb, depth, offsets_rgb, channels_rgb,
      img_size, normalize, x, y);
}

__global__ void ck_compute_depth_feature(RGBDPatchFeature *features,
    const float *depth, const Vector4i *offsets_depth,
    Vector2i img_size, bool normalize)
{
  const int x = threadIdx.x + blockIdx.x * blockDim.x;
  const int y = threadIdx.y + blockIdx.y * blockDim.y;

  if (x >= img_size.x || y >= img_size.y)
    return;

  compute_depth_patch_feature(features, depth, offsets_depth, img_size, normalize, x, y);
}

RGBDPatchFeatureCalculator_CUDA::RGBDPatchFeatureCalculator_CUDA()
{
  m_offsetsRgb->UpdateDeviceFromHost();
  m_channelsRgb->UpdateDeviceFromHost();
  m_offsetsDepth->UpdateDeviceFromHost();
}

void RGBDPatchFeatureCalculator_CUDA::ComputeFeature(
    const ITMUChar4Image_CPtr &rgb_image, const ITMFloatImage_CPtr &depth_image,
    boost::shared_ptr<ORUtils::Image<RGBDPatchFeature> > &features_image) const
{
  const Vector4u *rgb = rgb_image->GetData(MEMORYDEVICE_CUDA);
  const float *depth = depth_image->GetData(MEMORYDEVICE_CUDA);

  const Vector4i *offsets_rgb = m_offsetsRgb->GetData(MEMORYDEVICE_CUDA);
  const uchar *channels_rgb = m_channelsRgb->GetData(MEMORYDEVICE_CUDA);
  const Vector4i *offsets_depth = m_offsetsDepth->GetData(MEMORYDEVICE_CUDA);

  features_image->ChangeDims(rgb_image->noDims);
  features_image->Clear();
  RGBDPatchFeature *features = features_image->GetData(MEMORYDEVICE_CUDA);

  dim3 blockSize(32, 32);
  dim3 gridSize((rgb_image->noDims.x + blockSize.x - 1) / blockSize.x,
      (rgb_image->noDims.y + blockSize.y - 1) / blockSize.y);

  ck_compute_colour_feature<<<gridSize, blockSize>>>(features, rgb, depth, offsets_rgb, channels_rgb,
      rgb_image->noDims, m_normalizeRgb);
  hipDeviceSynchronize();

  ck_compute_depth_feature<<<gridSize, blockSize>>>(features, depth, offsets_depth, depth_image->noDims, m_normalizeDepth);
  hipDeviceSynchronize();
}

}
