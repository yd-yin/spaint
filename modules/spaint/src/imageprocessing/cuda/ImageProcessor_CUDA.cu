#include "hip/hip_runtime.h"
/**
 * spaint: ImageProcessor_CUDA.cu
 * Copyright (c) Torr Vision Group, University of Oxford, 2015. All rights reserved.
 */

#include "imageprocessing/cuda/ImageProcessor_CUDA.h"

#include "imageprocessing/shared/ImageProcessor_Shared.h"

namespace spaint {

//#################### CUDA KERNELS ####################

/**
 * \brief Calculates the pixel-wise absolute difference between two depth images.
 *
 * \param firstInputData   The data for the first input image (in row-major format).
 * \param secondInputData  The data for the second input image (in row-major format).
 * \param width            The width of each image.
 * \param height           The height of each image.
 * \param outputData       The location in which to store the result of the calculation (in column-major format).
 */
__global__ void ck_calculate_depth_difference(const float *firstInputData, const float *secondInputData, int width, int height, float *outputData)
{
  int tid = threadIdx.x + blockDim.x * blockIdx.x;
  if(tid < width * height)
  {
    calculate_pixel_depth_difference(tid, firstInputData, secondInputData, width, height, outputData);
  }
}

template <typename AFElementType, typename ITMElementType>
__global__ void ck_copy_af_to_itm(const AFElementType *inputData, int width, int height, ITMElementType *outputData)
{
  int tid = threadIdx.x + blockDim.x * blockIdx.x;
  if(tid < width * height)
  {
    copy_af_pixel_to_itm(tid, inputData, width, height, outputData);
  }
}

template <typename ITMElementType, typename AFElementType>
__global__ void ck_copy_itm_to_af(const ITMElementType *inputData, int width, int height, AFElementType *outputData)
{
  int tid = threadIdx.x + blockDim.x * blockIdx.x;
  if(tid < width * height)
  {
    copy_itm_pixel_to_af(tid, inputData, width, height, outputData);
  }
}

__global__ void ck_set_on_threshold(const float *inputData, int pixelCount, ImageProcessor::ComparisonOperator op, float threshold, float value, float *outputData)
{
  int tid = threadIdx.x + blockDim.x * blockIdx.x;
  if(tid < pixelCount)
  {
    set_pixel_on_threshold(tid, inputData, op, threshold, value, outputData);
  }
}

//#################### HELPER FUNCTIONS ####################

/**
 * \brief Copies an ArrayFire image to an InfiniTAM image using CUDA.
 *
 * \param inputImage  The input image.
 * \param outputImage The output image.
 */
template <typename AFElementType, typename ITMElementType>
static void copy_af_to_itm_cuda(const boost::shared_ptr<const af::array>& inputImage, const boost::shared_ptr<ORUtils::Image<ITMElementType> >& outputImage)
{
  const int height = outputImage->noDims.y;
  const int width = outputImage->noDims.x;
  const int pixelCount = height * width;

  int threadsPerBlock = 256;
  int numBlocks = (pixelCount + threadsPerBlock - 1) / threadsPerBlock;
  ck_copy_af_to_itm<<<numBlocks,threadsPerBlock>>>(
    inputImage->device<AFElementType>(),
    width,
    height,
    outputImage->GetData(MEMORYDEVICE_CUDA)
  );
  ORcudaKernelCheck;

  inputImage->unlock();
}

/**
 * \brief Copies an InfiniTAM image to an ArrayFire image using CUDA.
 *
 * \param inputImage  The input image.
 * \param outputImage The output image.
 */
template <typename ITMElementType, typename AFElementType>
static void copy_itm_to_af_cuda(const boost::shared_ptr<const ORUtils::Image<ITMElementType> >& inputImage, const boost::shared_ptr<af::array>& outputImage)
{
  const int height = inputImage->noDims.y;
  const int width = inputImage->noDims.x;
  const int pixelCount = height * width;

  int threadsPerBlock = 256;
  int numBlocks = (pixelCount + threadsPerBlock - 1) / threadsPerBlock;

  ck_copy_itm_to_af<<<numBlocks,threadsPerBlock>>>(
    inputImage->GetData(MEMORYDEVICE_CUDA),
    width,
    height,
    outputImage->device<AFElementType>()
  );
  ORcudaKernelCheck;

  outputImage->unlock();
}

//#################### PUBLIC MEMBER FUNCTIONS ####################

void ImageProcessor_CUDA::calculate_depth_difference(const ITMFloatImage_CPtr& firstInputImage, const ITMFloatImage_CPtr& secondInputImage, const AFArray_Ptr& outputImage) const
{
  check_image_size_equal(firstInputImage, secondInputImage);
  check_image_size_equal(firstInputImage, outputImage);

  Vector2i imgSize = image_size(outputImage);
  int pixelCount = imgSize.x * imgSize.y;

  int threadsPerBlock = 256;
  int numBlocks = (pixelCount + threadsPerBlock - 1) / threadsPerBlock;
  ck_calculate_depth_difference<<<numBlocks,threadsPerBlock>>>(
    firstInputImage->GetData(MEMORYDEVICE_CUDA),
    secondInputImage->GetData(MEMORYDEVICE_CUDA),
    imgSize.x,
    imgSize.y,
    outputImage->device<float>()
  );
  ORcudaKernelCheck;
}

void ImageProcessor_CUDA::copy_af_to_itm(const AFArray_CPtr& inputImage, const ITMFloatImage_Ptr& outputImage) const
{
  check_image_size_equal(inputImage, outputImage);
  copy_af_to_itm_cuda<float,float>(inputImage, outputImage);
}

void ImageProcessor_CUDA::copy_af_to_itm(const AFArray_CPtr& inputImage, const ITMUCharImage_Ptr& outputImage) const
{
  check_image_size_equal(inputImage, outputImage);
  copy_af_to_itm_cuda<unsigned char,unsigned char>(inputImage, outputImage);
}

void ImageProcessor_CUDA::copy_af_to_itm(const AFArray_CPtr& inputImage, const ITMUChar4Image_Ptr& outputImage) const
{
  check_image_size_equal(inputImage, outputImage);
  copy_af_to_itm_cuda<unsigned char,Vector4u>(inputImage, outputImage);
}

void ImageProcessor_CUDA::copy_itm_to_af(const ITMFloatImage_CPtr& inputImage, const AFArray_Ptr& outputImage) const
{
  check_image_size_equal(inputImage, outputImage);
  copy_itm_to_af_cuda<float,float>(inputImage, outputImage);
}

void ImageProcessor_CUDA::copy_itm_to_af(const ITMUCharImage_CPtr& inputImage, const AFArray_Ptr& outputImage) const
{
  check_image_size_equal(inputImage, outputImage);
  copy_itm_to_af_cuda<unsigned char,unsigned char>(inputImage, outputImage);
}

void ImageProcessor_CUDA::copy_itm_to_af(const ITMUChar4Image_CPtr& inputImage, const AFArray_Ptr& outputImage) const
{
  check_image_size_equal(inputImage, outputImage);
  copy_itm_to_af_cuda<Vector4u,unsigned char>(inputImage, outputImage);
}

void ImageProcessor_CUDA::set_on_threshold(const ITMFloatImage_CPtr& inputImage, ComparisonOperator op, float threshold, float value, const ITMFloatImage_Ptr& outputImage) const
{
  check_image_size_equal(inputImage, outputImage);

  Vector2i imgSize = inputImage->noDims;
  int pixelCount = imgSize.x * imgSize.y;

  int threadsPerBlock = 256;
  int numBlocks = (pixelCount + threadsPerBlock - 1) / threadsPerBlock;
  ck_set_on_threshold<<<numBlocks,threadsPerBlock>>>(
    inputImage->GetData(MEMORYDEVICE_CUDA),
    pixelCount,
    op,
    threshold,
    value,
    outputImage->GetData(MEMORYDEVICE_CUDA)
  );
  ORcudaKernelCheck;
}

}
