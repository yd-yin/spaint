#include "hip/hip_runtime.h"
/**
 * spaint: ImageProcessor_CUDA.cu
 */

#include "imageprocessing/cuda/ImageProcessor_CUDA.h"

#include "imageprocessing/shared/ImageProcessor_Shared.h"

namespace spaint {

//#################### CUDA KERNELS ####################

/**
 * \brief Calculates the pixel-wise absolute difference between two depth images.
 *
 * \param firstInputData   The data for the first input image (in row-major format).
 * \param secondInputData  The data for the second input image (in row-major format).
 * \param width            The width of each image.
 * \param height           The height of each image.
 * \param outputData       The location in which to store the result of the calculation (in column-major format).
 */
__global__ void ck_calculate_depth_difference(const float *firstInputData, const float *secondInputData, int width, int height, float *outputData)
{
  int tid = threadIdx.x + blockDim.x * blockIdx.x;
  if(tid < width * height)
  {
    calculate_pixel_depth_difference(tid, firstInputData, secondInputData, width, height, outputData);
  }
}

__global__ void ck_copy_af_to_itm(const unsigned char *inputData, int width, int height, unsigned char *outputData)
{
  int tid = threadIdx.x + blockDim.x * blockIdx.x;
  if(tid < width * height)
  {
    copy_af_pixel_to_itm(tid, inputData, width, height, outputData);
  }
}

__global__ void ck_set_on_threshold(const float *inputData, int pixelCount, ImageProcessor::ComparisonOperator op, float threshold, float value, float *outputData)
{
  int tid = threadIdx.x + blockDim.x * blockIdx.x;
  if(tid < pixelCount)
  {
    set_pixel_on_threshold(tid, inputData, op, threshold, value, outputData);
  }
}

//#################### PUBLIC MEMBER FUNCTIONS ####################

void ImageProcessor_CUDA::calculate_depth_difference(const ITMFloatImage_CPtr& firstInputImage, const ITMFloatImage_CPtr& secondInputImage, const AFArray_Ptr& outputImage) const
{
  check_image_size_equal(firstInputImage, secondInputImage);
  check_image_size_equal(firstInputImage, outputImage);

  Vector2i imgSize = image_size(outputImage);
  int pixelCount = imgSize.x * imgSize.y;

  int threadsPerBlock = 256;
  int numBlocks = (pixelCount + threadsPerBlock - 1) / threadsPerBlock;
  ck_calculate_depth_difference<<<numBlocks,threadsPerBlock>>>(
    firstInputImage->GetData(MEMORYDEVICE_CUDA),
    secondInputImage->GetData(MEMORYDEVICE_CUDA),
    imgSize.x,
    imgSize.y,
    outputImage->device<float>()
  );
}

void ImageProcessor_CUDA::copy_af_to_itm(const AFArray_CPtr& inputImage, const ITMUCharImage_Ptr& outputImage) const
{
  check_image_size_equal(inputImage, outputImage);
  
  Vector2i imgSize = outputImage->noDims;
  int pixelCount = imgSize.x * imgSize.y;

  int threadsPerBlock = 256;
  int numBlocks = (pixelCount + threadsPerBlock - 1) / threadsPerBlock;
  
  ck_copy_af_to_itm<<<numBlocks,threadsPerBlock>>>(
    inputImage->device<unsigned char>(),
    imgSize.x,
    imgSize.y,
    outputImage->GetData(MEMORYDEVICE_CUDA)
  );
}

void ImageProcessor_CUDA::set_on_threshold(const ITMFloatImage_CPtr& inputImage, ComparisonOperator op, float threshold, float value, const ITMFloatImage_Ptr& outputImage) const
{
  check_image_size_equal(inputImage, outputImage);

  Vector2i imgSize = inputImage->noDims;
  int pixelCount = imgSize.x * imgSize.y;

  int threadsPerBlock = 256;
  int numBlocks = (pixelCount + threadsPerBlock - 1) / threadsPerBlock;
  ck_set_on_threshold<<<numBlocks,threadsPerBlock>>>(
    inputImage->GetData(MEMORYDEVICE_CUDA),
    pixelCount,
    op,
    threshold,
    value,
    outputImage->GetData(MEMORYDEVICE_CUDA)
  );
}

}
