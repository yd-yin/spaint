#include "hip/hip_runtime.h"
/**
 * spaint: ImageProcessor_CUDA.cu
 */

#include "imageprocessing/cuda/ImageProcessor_CUDA.h"

#include "imageprocessing/shared/ImageProcessor_Shared.h"

namespace spaint {

//#################### CUDA KERNELS ####################

/**
 * \brief Calculates the pixel-wise absolute difference between two depth images.
 *
 * \param firstInputImage   The first input image (in row-major format).
 * \param secondInputImage  The second input image (in row-major format).
 * \param imgSize           The size of the images.
 * \param outputImage       The image in which to store the result of the calculation (in column-major format).
 */
__global__ void ck_calculate_depth_difference(const float *firstInputImage, const float *secondInputImage, Vector2i imgSize, float *outputImage)
{
  int x = blockIdx.x * blockDim.x + threadIdx.x, y = blockIdx.y * blockDim.y + threadIdx.y;
  if(x >= imgSize.x || y >= imgSize.y) return;

  int locIdcm = x * imgSize.y + y;
  int locIdrm = y * imgSize.x + x;
  calculate_pixel_depth_difference(firstInputImage[locIdrm], secondInputImage[locIdrm], &outputImage[locIdcm]);
}

__global__ void ck_set_on_threshold(float *output, const float *input, Vector2i imgSize, float threshold, ImageProcessor::ComparisonOperator op, float value)
{
  int x = blockIdx.x * blockDim.x + threadIdx.x, y = blockIdx.y * blockDim.y + threadIdx.y;
  if(x >= imgSize.x || y >= imgSize.y) return;

  int locId = y * imgSize.x + x;
  set_pixel_on_threshold(input[locId], op, threshold, value, &output[locId]);
}

//#################### PUBLIC MEMBER FUNCTIONS ####################

void ImageProcessor_CUDA::calculate_depth_difference(const ITMFloatImage_CPtr& firstInputImage, const ITMFloatImage_CPtr& secondInputImage, const AFImage_Ptr& outputImage) const
{
  check_image_size_equal(firstInputImage, secondInputImage);
  check_image_size_equal(firstInputImage, outputImage);

  Vector2i imgSize;
  imgSize.y = outputImage->dims(0);
  imgSize.x = outputImage->dims(1);

  dim3 cudaBlockSize(8, 8);
  dim3 gridSize((int)ceil((float)imgSize.x / (float)cudaBlockSize.x), (int)ceil((float)imgSize.y / (float)cudaBlockSize.y));
  ck_calculate_depth_difference<<<gridSize,cudaBlockSize>>>(
    firstInputImage->GetData(MEMORYDEVICE_CUDA),
    secondInputImage->GetData(MEMORYDEVICE_CUDA),
    imgSize,
    outputImage->device<float>()
  );
}

void ImageProcessor_CUDA::set_on_threshold(const ITMFloatImage_CPtr& input, ComparisonOperator op, float threshold, float value, const ITMFloatImage_Ptr& output) const
{
  check_image_size_equal(output, input);
  Vector2i imgSize = input->noDims;

  dim3 cudaBlockSize(8,8);
  dim3 gridSize((int)ceil((float)imgSize.x / (float)cudaBlockSize.x), (int)ceil((float)imgSize.y / (float)cudaBlockSize.y));
  ck_set_on_threshold<<<gridSize,cudaBlockSize>>>(
    output->GetData(MEMORYDEVICE_CUDA),
    input->GetData(MEMORYDEVICE_CUDA),
    imgSize,
    threshold,
    op,
    value
    );

}

}
