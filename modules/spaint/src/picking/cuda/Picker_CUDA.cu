#include "hip/hip_runtime.h"
/**
 * spaint: Picker_CUDA.cu
 */

#include "picking/cuda/Picker_CUDA.h"

#include "picking/shared/Picker_Shared.h"

namespace spaint {

//#################### CUDA KERNELS ####################

__global__ void ck_get_pick_point(int x, int y, int width, const Vector4f *imageData, Vector3f& pickPoint, bool& result)
{
  result = get_pick_point(x, y, width, imageData, pickPoint);
}

//#################### PUBLIC MEMBER FUNCTIONS ####################

bool Picker_CUDA::pick(int x, int y, const ITMLib::Objects::ITMRenderState *renderState, ORUtils::MemoryBlock<Vector3f>& pickPointMB) const
{
  static ORUtils::MemoryBlock<bool> result(1, true, true);
  ck_get_pick_point<<<1,1>>>(
    x, y,
    renderState->raycastResult->noDims.x,
    renderState->raycastResult->GetData(MEMORYDEVICE_CUDA),
    *pickPointMB.GetData(MEMORYDEVICE_CUDA),
    *result.GetData(MEMORYDEVICE_CUDA)
  );
  result.UpdateHostFromDevice();
  return result.GetData(MEMORYDEVICE_CPU)[0];
}

}
