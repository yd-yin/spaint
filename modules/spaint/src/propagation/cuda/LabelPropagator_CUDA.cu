#include "hip/hip_runtime.h"
/**
 * spaint: LabelPropagator_CUDA.cu
 * Copyright (c) Torr Vision Group, University of Oxford, 2015. All rights reserved.
 */

#include "propagation/cuda/LabelPropagator_CUDA.h"

#include "propagation/shared/LabelPropagator_Shared.h"

#define DEBUGGING 0

namespace spaint {

//#################### CUDA KERNELS ####################

__global__ void ck_calculate_normals(const Vector4f *raycastResultData, int raycastResultSize,
                                     const SpaintVoxel *voxelData, const ITMVoxelIndex::IndexData *indexData,
                                     Vector3f *surfaceNormals)
{
  int voxelIndex = threadIdx.x + blockDim.x * blockIdx.x;
  if(voxelIndex < raycastResultSize)
  {
    write_surface_normal(voxelIndex, raycastResultData, voxelData, indexData, surfaceNormals);
  }
}

__global__ void ck_perform_propagation(SpaintVoxel::Label label, const Vector4f *raycastResultData, int raycastResultSize, int width, int height,
                                       const Vector3f *surfaceNormals, SpaintVoxel *voxelData, const ITMVoxelIndex::IndexData *indexData,
                                       float maxAngleBetweenNormals, float maxSquaredDistanceBetweenColours, float maxSquaredDistanceBetweenVoxels)
{
  int voxelIndex = threadIdx.x + blockDim.x * blockIdx.x;
  if(voxelIndex < raycastResultSize)
  {
    propagate_from_neighbours(
      voxelIndex, width, height, label, raycastResultData, surfaceNormals, voxelData, indexData,
      maxAngleBetweenNormals, maxSquaredDistanceBetweenColours, maxSquaredDistanceBetweenVoxels
    );
  }
}

__global__ void ck_smooth_from_neighbours(const Vector4f *raycastResultData, int raycastResultSize, int width, int height, int maxLabelCount,
                                          SpaintVoxel *voxelData, const ITMVoxelIndex::IndexData *indexData)
{
  int voxelIndex = threadIdx.x + blockDim.x * blockIdx.x;
  if(voxelIndex < raycastResultSize)
  {
    smooth_from_neighbours(voxelIndex, width, height, maxLabelCount, raycastResultData, voxelData, indexData);
  }
}

//#################### CONSTRUCTORS ####################

LabelPropagator_CUDA::LabelPropagator_CUDA(size_t raycastResultSize, float maxAngleBetweenNormals, float maxSquaredDistanceBetweenColours, float maxSquaredDistanceBetweenVoxels)
: LabelPropagator(raycastResultSize, maxAngleBetweenNormals, maxSquaredDistanceBetweenColours, maxSquaredDistanceBetweenVoxels)
{}

//#################### PUBLIC MEMBER FUNCTIONS ####################

void LabelPropagator_CUDA::smooth_labels(const ITMFloat4Image *raycastResult, ITMLib::Objects::ITMScene<SpaintVoxel,ITMVoxelIndex> *scene) const
{
  const int raycastResultSize = static_cast<int>(raycastResult->dataSize);

  int threadsPerBlock = 256;
  int numBlocks = (raycastResultSize + threadsPerBlock - 1) / threadsPerBlock;

  const int maxLabelCount = 10; // TEMPORARY

  ck_smooth_from_neighbours<<<numBlocks,threadsPerBlock>>>(
    raycastResult->GetData(MEMORYDEVICE_CUDA),
    raycastResultSize,
    raycastResult->noDims.x,
    raycastResult->noDims.y,
    maxLabelCount,
    scene->localVBA.GetVoxelBlocks(),
    scene->index.getIndexData()
  );
}

//#################### PRIVATE MEMBER FUNCTIONS ####################

void LabelPropagator_CUDA::calculate_normals(const ITMFloat4Image *raycastResult, const ITMLib::Objects::ITMScene<SpaintVoxel,ITMVoxelIndex> *scene) const
{
  const int raycastResultSize = static_cast<int>(raycastResult->dataSize);

  int threadsPerBlock = 256;
  int numBlocks = (raycastResultSize + threadsPerBlock - 1) / threadsPerBlock;

  ck_calculate_normals<<<numBlocks,threadsPerBlock>>>(
    raycastResult->GetData(MEMORYDEVICE_CUDA),
    raycastResultSize,
    scene->localVBA.GetVoxelBlocks(),
    scene->index.getIndexData(),
    m_surfaceNormalsMB->GetData(MEMORYDEVICE_CUDA)
  );

#if DEBUGGING
  m_surfaceNormalsMB->UpdateHostFromDevice();
#endif
}

void LabelPropagator_CUDA::perform_propagation(SpaintVoxel::Label label, const ITMFloat4Image *raycastResult,
                                               ITMLib::Objects::ITMScene<SpaintVoxel,ITMVoxelIndex> *scene) const
{
  const int raycastResultSize = static_cast<int>(raycastResult->dataSize);

  int threadsPerBlock = 256;
  int numBlocks = (raycastResultSize + threadsPerBlock - 1) / threadsPerBlock;

  ck_perform_propagation<<<numBlocks,threadsPerBlock>>>(
    label,
    raycastResult->GetData(MEMORYDEVICE_CUDA),
    raycastResultSize,
    raycastResult->noDims.x,
    raycastResult->noDims.y,
    m_surfaceNormalsMB->GetData(MEMORYDEVICE_CUDA),
    scene->localVBA.GetVoxelBlocks(),
    scene->index.getIndexData(),
    m_maxAngleBetweenNormals,
    m_maxSquaredDistanceBetweenColours,
    m_maxSquaredDistanceBetweenVoxels
  );
}

}
