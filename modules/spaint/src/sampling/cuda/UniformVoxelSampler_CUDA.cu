#include "hip/hip_runtime.h"
/**
 * spaint: UniformVoxelSampler_CUDA.cu
 */

#include "sampling/cuda/UniformVoxelSampler_CUDA.h"

#include "sampling/shared/UniformVoxelSampler_Shared.h"

namespace spaint {

//#################### CUDA KERNELS ####################

__global__ void ck_write_sampled_voxel_locations(size_t voxelsToSample, const Vector4f *raycastResultData, const int *sampledVoxelIndices, Vector3s *sampledVoxelLocations)
{
  int tid = threadIdx.x + blockDim.x * blockIdx.x;
  if(tid < static_cast<int>(voxelsToSample))
  {
    write_sampled_voxel_location(tid, raycastResultData, sampledVoxelIndices, sampledVoxelLocations);
  }
}

//#################### CONSTRUCTORS ####################

UniformVoxelSampler_CUDA::UniformVoxelSampler_CUDA(int raycastResultSize, unsigned int seed)
: UniformVoxelSampler(raycastResultSize, seed)
{}

//#################### PRIVATE MEMBER FUNCTIONS ####################

void UniformVoxelSampler_CUDA::write_sampled_voxel_locations(const ITMFloat4Image *raycastResult, size_t sampledVoxelCount,
                                                             ORUtils::MemoryBlock<Vector3s>& sampledVoxelLocationsMB) const
{
  int threadsPerBlock = 256;
  int numBlocks = (static_cast<int>(sampledVoxelCount) + threadsPerBlock - 1) / threadsPerBlock;

  ck_write_sampled_voxel_locations<<<numBlocks,threadsPerBlock>>>(
    sampledVoxelCount,
    raycastResult->GetData(MEMORYDEVICE_CUDA),
    m_sampledVoxelIndicesMB.GetData(MEMORYDEVICE_CUDA),
    sampledVoxelLocationsMB.GetData(MEMORYDEVICE_CUDA)
  );
}

}
