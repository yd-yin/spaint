#include "hip/hip_runtime.h"
/**
 * spaint: GPURansac_CUDA.cu
 * Copyright (c) Torr Vision Group, University of Oxford, 2016. All rights reserved.
 */

#include "randomforest/cuda/GPURansac_CUDA.h"

#include "util/MemoryBlockFactory.h"

#include <Eigen/Dense>

#include <thrust/device_ptr.h>
#include <thrust/execution_policy.h>
#include <thrust/sort.h>

namespace spaint
{

namespace
{
__global__ void ck_init_random_states(GPURansac_CUDA::RandomState *randomStates,
    uint32_t nbStates, uint32_t seed)
{
  int idx = threadIdx.x + blockIdx.x * blockDim.x;

  if (idx >= nbStates)
    return;

  hiprand_init(seed, idx, 0, &randomStates[idx]);
}

__global__ void ck_reset_pose_candidates(PoseCandidates *poseCandidates)
{
  poseCandidates->nbCandidates = 0;
}

__device__ bool generate_candidate(const RGBDPatchFeature *patchFeaturesData,
    const GPUForestPrediction *predictionsData, const Vector2i &imgSize,
    GPURansac_CUDA::RandomState *randomState, PoseCandidate &poseCandidate,
    bool m_useAllModesPerLeafInPoseHypothesisGeneration,
    bool m_checkMinDistanceBetweenSampledModes,
    float m_minDistanceBetweenSampledModes,
    bool m_checkRigidTransformationConstraint,
    float m_translationErrorMaxForCorrectPose)
{
  static const int m_nbPointsForKabschBoostrap = 3;

//
//  std::uniform_int_distribution<int> col_index_generator(0,
//      m_featureImage->noDims.width - 1);
//  std::uniform_int_distribution<int> row_index_generator(0,
//      m_featureImage->noDims.height - 1);
//
//  const RGBDPatchFeature *patchFeaturesData = m_featureImage->GetData(
//      MEMORYDEVICE_CPU);
//  const GPUForestPrediction *predictionsData = m_predictionsImage->GetData(
//      MEMORYDEVICE_CPU);

  bool foundIsometricMapping = false;
  const int maxIterationsOuter = 20;
  int iterationsOuter = 0;

  while (!foundIsometricMapping && iterationsOuter < maxIterationsOuter)
  {
    ++iterationsOuter;

    int selectedPixelCount = 0;
    int selectedPixelX[m_nbPointsForKabschBoostrap];
    int selectedPixelY[m_nbPointsForKabschBoostrap];
    int selectedPixelMode[m_nbPointsForKabschBoostrap];

    static const int maxIterationsInner = 6000;
    int iterationsInner = 0;

    while (selectedPixelCount != m_nbPointsForKabschBoostrap
        && iterationsInner < maxIterationsInner)
    {
      ++iterationsInner;

      const int x = hiprand(randomState) % (imgSize.width - 1);
      const int y = hiprand(randomState) % (imgSize.height - 1);
      // The implementation below sometimes generates OOB values (with 0.999999,
      // with 0.999 it works but seems a weird hack)
//      const int x = __float2int_rz(
//          hiprand_uniform(randomState) * (imgSize.width - 1 + 0.999999f));
//      const int y = __float2int_rz(
//          hiprand_uniform(randomState) * (imgSize.height - 1 + 0.999999f));
      const int linearFeatureIdx = y * imgSize.width + x;
      const RGBDPatchFeature &selectedFeature =
          patchFeaturesData[linearFeatureIdx];

      if (selectedFeature.position.w < 0.f) // Invalid feature
        continue;

      const GPUForestPrediction &selectedPrediction =
          predictionsData[linearFeatureIdx];

      if (selectedPrediction.nbModes == 0)
        continue;

      int selectedModeIdx = 0;
      if (m_useAllModesPerLeafInPoseHypothesisGeneration)
      {
        selectedModeIdx = hiprand(randomState)
            % (selectedPrediction.nbModes - 1);
//        selectedModeIdx = __float2int_rz(
//            hiprand_uniform(randomState)
//                * (selectedPrediction.nbModes - 1 + 0.999f));
      }

      // This is the first pixel, check that the pixel colour corresponds with the selected mode
      if (selectedPixelCount == 0)
      {
        const Vector3u colourDiff = selectedFeature.colour.toVector3().toUChar()
            - selectedPrediction.modes[selectedModeIdx].colour;
        const bool consistentColour = fabsf(colourDiff.x) <= 30.f
            && fabsf(colourDiff.y) <= 30.f && fabsf(colourDiff.z) <= 30.f;

        if (!consistentColour)
          continue;
      }

      // if (false)
      if (m_checkMinDistanceBetweenSampledModes)
      {
        const Vector3f worldPt =
            selectedPrediction.modes[selectedModeIdx].position;

        // Check that this mode is far enough from the other modes
        bool farEnough = true;

        for (int idxOther = 0; idxOther < selectedPixelCount; ++idxOther)
        {
          const int xOther = selectedPixelX[idxOther];
          const int yOther = selectedPixelY[idxOther];
          const int modeIdxOther = selectedPixelMode[idxOther];

          const int linearIdxOther = yOther * imgSize.width + xOther;
          const GPUForestPrediction &predOther = predictionsData[linearIdxOther];

          Vector3f worldPtOther = predOther.modes[modeIdxOther].position;

          float distOther = length(worldPtOther - worldPt);
          if (distOther < m_minDistanceBetweenSampledModes)
          {
            farEnough = false;
            break;
          }
        }

        if (!farEnough)
          continue;
      }

      // isometry?
      //       if (false)
      // if (true)
      if (m_checkRigidTransformationConstraint)
      {
        bool violatesConditions = false;

        for (int m = 0; m < selectedPixelCount && !violatesConditions; ++m)
        {
          const int xFirst = selectedPixelX[m];
          const int yFirst = selectedPixelY[m];
          const int modeIdxFirst = selectedPixelMode[m];
          const int linearIdxOther = yFirst * imgSize.width + xFirst;
          const GPUForestPrediction &predFirst = predictionsData[linearIdxOther];

          const Vector3f worldPtFirst = predFirst.modes[modeIdxFirst].position;
          const Vector3f worldPtCur =
              selectedPrediction.modes[selectedModeIdx].position;

          float distWorld = length(worldPtFirst - worldPtCur);

          const Vector3f localPred =
              patchFeaturesData[linearIdxOther].position.toVector3();
          const Vector3f localCur = selectedFeature.position.toVector3();

          float distLocal = length(localPred - localCur);

          if (distLocal < m_minDistanceBetweenSampledModes)
            violatesConditions = true;

          if (std::abs(distLocal - distWorld)
              > 0.5f * m_translationErrorMaxForCorrectPose)
          {
            violatesConditions = true;
          }
        }

        if (violatesConditions)
          continue;
      }

      selectedPixelX[selectedPixelCount] = x;
      selectedPixelY[selectedPixelCount] = y;
      selectedPixelMode[selectedPixelCount] = selectedModeIdx;
      ++selectedPixelCount;
    }

    //    std::cout << "Inner iterations: " << iterationsInner << std::endl;

    // Reached limit of iterations
    if (selectedPixelCount != m_nbPointsForKabschBoostrap)
      return false;

    // Populate resulting pose (except the actual pose that is computed on the CPU due to Kabsch)
    foundIsometricMapping = true;
    poseCandidate.nbInliers = selectedPixelCount;
    poseCandidate.energy = 0.f;
    poseCandidate.cameraId = -1;

    for (int s = 0; s < selectedPixelCount; ++s)
    {
      const int x = selectedPixelX[s];
      const int y = selectedPixelY[s];
      const int modeIdx = selectedPixelMode[s];
      const int linearIdx = y * imgSize.width + x;

      poseCandidate.inliers[s].linearIdx = linearIdx;
      poseCandidate.inliers[s].modeIdx = modeIdx;
      poseCandidate.inliers[s].energy = 0.f;
    }
  }

  if (iterationsOuter < maxIterationsOuter)
    return true;

  return false;
}

__global__ void ck_generate_pose_candidates(const RGBDPatchFeature *features,
    const GPUForestPrediction *predictions, const Vector2i imgSize,
    GPURansac_CUDA::RandomState *randomStates, PoseCandidate *poseCandidates,
    int *nbPoseCandidates, int maxNbPoseCandidates,
    bool m_useAllModesPerLeafInPoseHypothesisGeneration,
    bool m_checkMinDistanceBetweenSampledModes,
    float m_minDistanceBetweenSampledModes,
    bool m_checkRigidTransformationConstraint,
    float m_translationErrorMaxForCorrectPose)
{
  const int candidateIdx = blockIdx.x * blockDim.x + threadIdx.x;

  if (candidateIdx >= maxNbPoseCandidates)
    return;

  GPURansac_CUDA::RandomState *randomState = &randomStates[candidateIdx];

  PoseCandidate candidate;
  candidate.cameraId = candidateIdx;

  bool valid = generate_candidate(features, predictions, imgSize, randomState,
      candidate, m_useAllModesPerLeafInPoseHypothesisGeneration,
      m_checkMinDistanceBetweenSampledModes, m_minDistanceBetweenSampledModes,
      m_checkRigidTransformationConstraint,
      m_translationErrorMaxForCorrectPose);

  if (valid)
  {
    const int candidateIdx = atomicAdd(nbPoseCandidates, 1);
    poseCandidates[candidateIdx] = candidate;
  }
}

__global__ void ck_compute_energies(const RGBDPatchFeature *features,
    const GPUForestPrediction *predictions, PoseCandidate *poseCandidates,
    int nbCandidates)
{
  const int tId = threadIdx.x;
  const int threadsPerBlock = blockDim.x;
  const int candidateIdx = blockIdx.x;

  if (candidateIdx >= nbCandidates)
  {
    // Candidate has been trimmed, entire block returns,
    // does not cause troubles with the following __syncthreads()
    return;
  }

  PoseCandidate &currentCandidate = poseCandidates[candidateIdx];

  float localEnergy = 0.f;

  const int nbInliers = currentCandidate.nbInliers;
  for (int inlierIdx = tId; inlierIdx < nbInliers; inlierIdx += threadsPerBlock)
  {
    const int linearIdx = currentCandidate.inliers[inlierIdx].linearIdx;
    const Vector3f localPixel = features[linearIdx].position.toVector3();
    const Vector3f projectedPixel = currentCandidate.cameraPose * localPixel;

    const GPUForestPrediction &pred = predictions[linearIdx];

    // eval individual energy
    float energy;
    int argmax = pred.get_best_mode_and_energy(projectedPixel, energy);

    // Has at least a valid mode
    if (argmax < 0)
    {
      // should have not been inserted in the inlier set
      printf("prediction has no valid modes\n");
      continue;
    }

    if (pred.modes[argmax].nbInliers == 0)
    {
      // the original implementation had a simple continue
      printf("mode has no inliers\n");
      continue;
    }

    energy /= static_cast<float>(pred.nbModes);
    energy /= static_cast<float>(pred.modes[argmax].nbInliers);

    if (energy < 1e-6f)
      energy = 1e-6f;
    energy = -log10f(energy);

    currentCandidate.inliers[inlierIdx].energy = energy;
    currentCandidate.inliers[inlierIdx].modeIdx = argmax;
    localEnergy += energy;
  }

  // Now reduce by shuffling down the local energies
  //(localEnergy for thread 0 in the warp contains the sum for the warp)
  for (int offset = warpSize / 2; offset > 0; offset /= 2)
    localEnergy += __shfl_down(localEnergy, offset);

  // Thread 0 of each warp updates the final energy
  if ((threadIdx.x & (warpSize - 1)) == 0)
    atomicAdd(&currentCandidate.energy, localEnergy);

  __syncthreads(); // Wait for all threads in the block

  // tId 0 computes the final energy
  if (tId == 0)
    currentCandidate.energy = currentCandidate.energy
        / static_cast<float>(currentCandidate.nbInliers);
}

__global__ void ck_reset_candidate_energies(PoseCandidate *poseCandidates,
    int nbPoseCandidates)
{
  const int candidateIdx = threadIdx.x;

  if (candidateIdx >= nbPoseCandidates)
  {
    return;
  }

  poseCandidates[candidateIdx].energy = 0.f;
}
}

__device__ bool compare_poses_ascending_energy(const PoseCandidate &a,
    const PoseCandidate &b)
{
  return a.energy < b.energy;
}

GPURansac_CUDA::GPURansac_CUDA() :
    GPURansac()
{
  MemoryBlockFactory &mbf = MemoryBlockFactory::instance();
  m_randomStates = mbf.make_block<RandomState>(PoseCandidates::MAX_CANDIDATES);
  m_rngSeed = 42;
  m_nbPoseCandidates_device = mbf.make_image<int>(Vector2i(1,1));

  init_random();
}

void GPURansac_CUDA::init_random()
{
  RandomState *randomStates = m_randomStates->GetData(MEMORYDEVICE_CUDA);

  // Initialize random states
  dim3 blockSize(256);
  dim3 gridSize(
      (PoseCandidates::MAX_CANDIDATES + blockSize.x - 1) / blockSize.x);

  ck_init_random_states<<<gridSize, blockSize>>>(randomStates, PoseCandidates::MAX_CANDIDATES, m_rngSeed);
}

void GPURansac_CUDA::generate_pose_candidates()
{
  const Vector2i imgSize = m_featureImage->noDims;
  const RGBDPatchFeature *features = m_featureImage->GetData(MEMORYDEVICE_CUDA);
  const GPUForestPrediction *predictions = m_predictionsImage->GetData(
      MEMORYDEVICE_CUDA);

  RandomState *randomStates = m_randomStates->GetData(MEMORYDEVICE_CUDA);
  PoseCandidate *poseCandidates = m_poseCandidates->GetData(MEMORYDEVICE_CUDA);
  int *nbPoseCandidates = m_nbPoseCandidates_device->GetData(MEMORYDEVICE_CUDA);

  dim3 blockSize(32);
  dim3 gridSize(
      (PoseCandidates::MAX_CANDIDATES + blockSize.x - 1) / blockSize.x);

  // Reset number of candidates (only on device, the host number will be updated later)
  ORcudaSafeCall(hipMemsetAsync(nbPoseCandidates, 0, sizeof(int)));

  ck_generate_pose_candidates<<<gridSize, blockSize>>>(features, predictions, imgSize, randomStates,
      poseCandidates, nbPoseCandidates, PoseCandidates::MAX_CANDIDATES,
      m_useAllModesPerLeafInPoseHypothesisGeneration,
      m_checkMinDistanceBetweenSampledModes, m_minDistanceBetweenSampledModes,
      m_checkRigidTransformationConstraint,
      m_translationErrorMaxForCorrectPose);
  ORcudaKernelCheck;

  // Need to make the data available to the host
  m_poseCandidates->UpdateHostFromDevice();
  m_nbPoseCandidates = m_nbPoseCandidates_device->GetElement(0, MEMORYDEVICE_CUDA);

  // Now perform kabsch on the candidates
  //#ifdef ENABLE_TIMERS
  //    boost::timer::auto_cpu_timer t(6,
  //        "kabsch: %ws wall, %us user + %ss system = %ts CPU (%p%)\n");
  //#endif
  compute_candidate_pose_kabsch();
}

void GPURansac_CUDA::compute_and_sort_energies()
{
  GPURansac::compute_and_sort_energies();
  return;

//  // Need to make the data available to the device
//  m_poseCandidates->UpdateDeviceFromHost();
//  m_nbPoseCandidates->UpdateDeviceFromHost();
//
//  const RGBDPatchFeature *features = m_featureImage->GetData(MEMORYDEVICE_CUDA);
//  const GPUForestPrediction *predictions = m_predictionsImage->GetData(
//      MEMORYDEVICE_CUDA);
//  PoseCandidate *poseCandidates = m_poseCandidates->GetData(MEMORYDEVICE_CUDA);
//  const int nbPoseCandidates = *m_nbPoseCandidates->GetData(MEMORYDEVICE_CPU);
//
//  ck_reset_candidate_energies<<<1, nbPoseCandidates>>>(poseCandidates, nbPoseCandidates);
//  ORcudaKernelCheck;
//
//  dim3 blockSize(128); // threads to compute the energy for each candidate
//  dim3 gridSize(nbPoseCandidates); // Launch one block per candidate (many blocks will exit immediately in the later stages of ransac)
//  ck_compute_energies<<<gridSize, blockSize>>>(features, predictions, poseCandidates, nbPoseCandidates);
//  ORcudaKernelCheck;
//
//  // Sort by ascending energy
//  thrust::device_ptr<PoseCandidate> candidatesStart(poseCandidates);
//  thrust::device_ptr<PoseCandidate> candidatesEnd(
//      poseCandidates + nbPoseCandidates);
//  thrust::sort(candidatesStart, candidatesEnd);
//
//  // Need to make the data available to the host once again
//  m_poseCandidates->UpdateHostFromDevice();
}

void GPURansac_CUDA::compute_candidate_pose_kabsch()
{
  const RGBDPatchFeature *features = m_featureImage->GetData(MEMORYDEVICE_CPU);
  const GPUForestPrediction *predictions = m_predictionsImage->GetData(
      MEMORYDEVICE_CPU);
  PoseCandidate *poseCandidates = m_poseCandidates->GetData(MEMORYDEVICE_CPU);

//  std::cout << "Generated " << nbPoseCandidates << " candidates." << std::endl;

#pragma omp parallel for
  for (int candidateIdx = 0; candidateIdx < m_nbPoseCandidates; ++candidateIdx)
  {
    PoseCandidate &candidate = poseCandidates[candidateIdx];

    Eigen::MatrixXf localPoints(3, candidate.nbInliers);
    Eigen::MatrixXf worldPoints(3, candidate.nbInliers);
    for (int s = 0; s < candidate.nbInliers; ++s)
    {
      const int linearIdx = candidate.inliers[s].linearIdx;
      const int modeIdx = candidate.inliers[s].modeIdx;
      const GPUForestPrediction &pred = predictions[linearIdx];

      localPoints.col(s) = Eigen::Map<const Eigen::Vector3f>(
          features[linearIdx].position.v);
      worldPoints.col(s) = Eigen::Map<const Eigen::Vector3f>(
          pred.modes[modeIdx].position.v);
    }

    Eigen::Map<Eigen::Matrix4f>(candidate.cameraPose.m) = Kabsch(localPoints,
        worldPoints);
  }
}

}
