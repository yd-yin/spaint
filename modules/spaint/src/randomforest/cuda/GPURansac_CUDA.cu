#include "hip/hip_runtime.h"
/**
 * spaint: GPURansac_CUDA.cu
 * Copyright (c) Torr Vision Group, University of Oxford, 2016. All rights reserved.
 */

#include "randomforest/cuda/GPURansac_CUDA.h"

#include "util/MemoryBlockFactory.h"

namespace spaint
{

namespace
{
__global__ void ck_init_random_states(GPURansac_CUDA::RandomState *randomStates,
    uint32_t nbStates, uint32_t seed)
{
  int idx = threadIdx.x + blockIdx.x * blockDim.x;

  if (idx >= nbStates)
    return;

  hiprand_init(seed, idx, 0, &randomStates[idx]);
}

__device__ bool generate_candidate(const RGBDPatchFeature *patchFeaturesData,
    const GPUForestPrediction *predictionsData, const Vector2i &imgSize,
    GPURansac_CUDA::RandomState *randomState, PoseCandidate &poseCandidate,
    bool m_useAllModesPerLeafInPoseHypothesisGeneration,
    bool m_checkMinDistanceBetweenSampledModes,
    float m_minDistanceBetweenSampledModes,
    bool m_checkRigidTransformationConstraint,
    float m_translationErrorMaxForCorrectPose)
{
  static const int m_nbPointsForKabschBoostrap = 3;

//
//  std::uniform_int_distribution<int> col_index_generator(0,
//      m_featureImage->noDims.width - 1);
//  std::uniform_int_distribution<int> row_index_generator(0,
//      m_featureImage->noDims.height - 1);
//
//  const RGBDPatchFeature *patchFeaturesData = m_featureImage->GetData(
//      MEMORYDEVICE_CPU);
//  const GPUForestPrediction *predictionsData = m_predictionsImage->GetData(
//      MEMORYDEVICE_CPU);

  bool foundIsometricMapping = false;
  const int maxIterationsOuter = 20;
  int iterationsOuter = 0;

  while (!foundIsometricMapping && iterationsOuter < maxIterationsOuter)
  {
    ++iterationsOuter;

    int selectedPixelCount = 0;
    int selectedPixelX[m_nbPointsForKabschBoostrap];
    int selectedPixelY[m_nbPointsForKabschBoostrap];
    int selectedPixelMode[m_nbPointsForKabschBoostrap];

    static const int maxIterationsInner = 6000;
    int iterationsInner = 0;

    while (selectedPixelCount != m_nbPointsForKabschBoostrap
        && iterationsInner < maxIterationsInner)
    {
      ++iterationsInner;

      const int x = hiprand(randomState) % (imgSize.width - 1);
      const int y = hiprand(randomState) % (imgSize.height - 1);
      // The implementation below sometimes generates OOB values (with 0.999999,
      // with 0.999 it works but seems a weird hack)
//      const int x = __float2int_rz(
//          hiprand_uniform(randomState) * (imgSize.width - 1 + 0.999999f));
//      const int y = __float2int_rz(
//          hiprand_uniform(randomState) * (imgSize.height - 1 + 0.999999f));
      const int linearFeatureIdx = y * imgSize.width + x;
      const RGBDPatchFeature &selectedFeature =
          patchFeaturesData[linearFeatureIdx];

      if (selectedFeature.position.w < 0.f) // Invalid feature
        continue;

      const GPUForestPrediction &selectedPrediction =
          predictionsData[linearFeatureIdx];

      if (selectedPrediction.nbModes == 0)
        continue;

      int selectedModeIdx = 0;
      if (m_useAllModesPerLeafInPoseHypothesisGeneration)
      {
        selectedModeIdx = hiprand(randomState)
            % (selectedPrediction.nbModes - 1);
//        selectedModeIdx = __float2int_rz(
//            hiprand_uniform(randomState)
//                * (selectedPrediction.nbModes - 1 + 0.999f));
      }

      // This is the first pixel, check that the pixel colour corresponds with the selected mode
      if (selectedPixelCount == 0)
      {
        const Vector3u colourDiff = selectedFeature.colour.toVector3().toUChar()
            - selectedPrediction.modes[selectedModeIdx].colour;
        const bool consistentColour = fabsf(colourDiff.x) <= 30.f
            && fabsf(colourDiff.y) <= 30.f && fabsf(colourDiff.z) <= 30.f;

        if (!consistentColour)
          continue;
      }

      // if (false)
      if (m_checkMinDistanceBetweenSampledModes)
      {
        const Vector3f worldPt =
            selectedPrediction.modes[selectedModeIdx].position;

        // Check that this mode is far enough from the other modes
        bool farEnough = true;

        for (int idxOther = 0; idxOther < selectedPixelCount; ++idxOther)
        {
          const int xOther = selectedPixelX[idxOther];
          const int yOther = selectedPixelY[idxOther];
          const int modeIdxOther = selectedPixelMode[idxOther];

          const int linearIdxOther = yOther * imgSize.width + xOther;
          const GPUForestPrediction &predOther = predictionsData[linearIdxOther];

          Vector3f worldPtOther = predOther.modes[modeIdxOther].position;

          float distOther = length(worldPtOther - worldPt);
          if (distOther < m_minDistanceBetweenSampledModes)
          {
            farEnough = false;
            break;
          }
        }

        if (!farEnough)
          continue;
      }

      // isometry?
      //       if (false)
      // if (true)
      if (m_checkRigidTransformationConstraint)
      {
        bool violatesConditions = false;

        for (int m = 0; m < selectedPixelCount && !violatesConditions; ++m)
        {
          const int xFirst = selectedPixelX[m];
          const int yFirst = selectedPixelY[m];
          const int modeIdxFirst = selectedPixelMode[m];
          const int linearIdxOther = yFirst * imgSize.width + xFirst;
          const GPUForestPrediction &predFirst = predictionsData[linearIdxOther];

          const Vector3f worldPtFirst = predFirst.modes[modeIdxFirst].position;
          const Vector3f worldPtCur =
              selectedPrediction.modes[selectedModeIdx].position;

          float distWorld = length(worldPtFirst - worldPtCur);

          const Vector3f localPred =
              patchFeaturesData[linearIdxOther].position.toVector3();
          const Vector3f localCur = selectedFeature.position.toVector3();

          float distLocal = length(localPred - localCur);

          if (distLocal < m_minDistanceBetweenSampledModes)
            violatesConditions = true;

          if (std::abs(distLocal - distWorld)
              > 0.5f * m_translationErrorMaxForCorrectPose)
          {
            violatesConditions = true;
          }
        }

        if (violatesConditions)
          continue;
      }

      selectedPixelX[selectedPixelCount] = x;
      selectedPixelY[selectedPixelCount] = y;
      selectedPixelMode[selectedPixelCount] = selectedModeIdx;
      ++selectedPixelCount;
    }

    //    std::cout << "Inner iterations: " << iterationsInner << std::endl;

    // Reached limit of iterations
    if (selectedPixelCount != m_nbPointsForKabschBoostrap)
      return false;

    // Populate resulting pose (except the actual pose that is computed on the CPU due to Kabsch)
    foundIsometricMapping = true;
    poseCandidate.nbInliers = selectedPixelCount;
    poseCandidate.energy = 0.f;
    poseCandidate.cameraId = -1;

    for (int s = 0; s < selectedPixelCount; ++s)
    {
      const int x = selectedPixelX[s];
      const int y = selectedPixelY[s];
      const int modeIdx = selectedPixelMode[s];
      const int linearIdx = y * imgSize.width + x;

      poseCandidate.inliers[s].linearIdx = linearIdx;
      poseCandidate.inliers[s].modeIdx = modeIdx;
      poseCandidate.inliers[s].energy = 0.f;
    }
  }

  if (iterationsOuter < maxIterationsOuter)
    return true;

  return false;
}

__global__ void ck_generate_pose_candidates(const RGBDPatchFeature *features,
    const GPUForestPrediction *predictions, const Vector2i imgSize,
    GPURansac_CUDA::RandomState *randomStates, PoseCandidates *poseCandidates,
    int maxNbPoseCandidates,
    bool m_useAllModesPerLeafInPoseHypothesisGeneration,
    bool m_checkMinDistanceBetweenSampledModes,
    float m_minDistanceBetweenSampledModes,
    bool m_checkRigidTransformationConstraint,
    float m_translationErrorMaxForCorrectPose)
{
  const int candidateIdx = blockIdx.x * blockDim.x + threadIdx.x;

  // Reset nuber of candidates. Might put in a different kernel for efficiency.
  if (candidateIdx == 0)
    poseCandidates->nbCandidates = 0;

  __syncthreads();

  if (candidateIdx >= maxNbPoseCandidates)
    return;

  GPURansac_CUDA::RandomState *randomState = &randomStates[candidateIdx];

  PoseCandidate candidate;
  candidate.cameraId = candidateIdx;

  bool valid = generate_candidate(features, predictions, imgSize, randomState,
      candidate, m_useAllModesPerLeafInPoseHypothesisGeneration,
      m_checkMinDistanceBetweenSampledModes, m_minDistanceBetweenSampledModes,
      m_checkRigidTransformationConstraint,
      m_translationErrorMaxForCorrectPose);

  if (valid)
  {
    const int candidateIdx = atomicAdd(&poseCandidates->nbCandidates, 1);

    PoseCandidate *candidates = poseCandidates->candidates;
    candidates[candidateIdx] = candidate;
  }
}
}

GPURansac_CUDA::GPURansac_CUDA()
{
  MemoryBlockFactory &mbf = MemoryBlockFactory::instance();
  m_randomStates = mbf.make_block<RandomState>(PoseCandidates::MAX_CANDIDATES);
  m_rngSeed = 42;

  init_random();
}

void GPURansac_CUDA::init_random()
{
  RandomState *randomStates = m_randomStates->GetData(MEMORYDEVICE_CUDA);

  // Initialize random states
  dim3 blockSize(256);
  dim3 gridSize(
      (PoseCandidates::MAX_CANDIDATES + blockSize.x - 1) / blockSize.x);

  ck_init_random_states<<<gridSize, blockSize>>>(randomStates, PoseCandidates::MAX_CANDIDATES, m_rngSeed);
}

void GPURansac_CUDA::generate_pose_candidates()
{
  const Vector2i imgSize = m_featureImage->noDims;
  const RGBDPatchFeature *features = m_featureImage->GetData(MEMORYDEVICE_CUDA);
  const GPUForestPrediction *predictions = m_predictionsImage->GetData(
      MEMORYDEVICE_CUDA);

  RandomState *randomStates = m_randomStates->GetData(MEMORYDEVICE_CUDA);
  PoseCandidates *poseCandidates = m_poseCandidates->GetData(MEMORYDEVICE_CUDA);

  dim3 blockSize(128);
  dim3 gridSize(
      (PoseCandidates::MAX_CANDIDATES + blockSize.x - 1) / blockSize.x);
  ck_generate_pose_candidates<<<gridSize, blockSize>>>(features, predictions, imgSize, randomStates,
      poseCandidates, PoseCandidates::MAX_CANDIDATES, m_useAllModesPerLeafInPoseHypothesisGeneration, m_checkMinDistanceBetweenSampledModes, m_minDistanceBetweenSampledModes, m_checkRigidTransformationConstraint, m_translationErrorMaxForCorrectPose);
  hipDeviceSynchronize();
}

}
