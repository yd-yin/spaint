#include "hip/hip_runtime.h"
/**
 * spaint: GPUForest_CUDA.cu
 * Copyright (c) Torr Vision Group, University of Oxford, 2016. All rights reserved.
 */

#include "randomforest/cuda/GPUForest_CUDA.h"

#include "randomforest/shared/GPUForest_Shared.h"

namespace spaint
{
__global__ void ck_evaluate_forest(const GPUForestNode* forestTexture,
    int nbTrees, const RGBDPatchFeature* featureData, Vector2i imgSize,
    int* leafData, Vector2i leafSize)
{
  const int x = blockIdx.x * blockDim.x + threadIdx.x;
  const int y = blockIdx.y * blockDim.y + threadIdx.y;

  if (x >= imgSize.x || y >= imgSize.y)
    return;

  evaluate_forest(forestTexture, nbTrees, featureData, imgSize, leafData, leafSize, x, y);
}

GPUForest_CUDA::GPUForest_CUDA(const EnsembleLearner &pretrained_forest) :
    GPUForest(pretrained_forest)
{
  m_forestImage->UpdateDeviceFromHost();
}

void GPUForest_CUDA::evaluate_forest(const RGBDPatchFeatureImage_CPtr &features,
    ITMIntImage_Ptr &leaf_indices) const
{
  const int nbTrees = m_forestImage->noDims.width;
  const GPUForestNode* forestTexture = m_forestImage->GetData(
      MEMORYDEVICE_CUDA);

  const Vector2i imgSize = features->noDims;
  const RGBDPatchFeature* featureData = features->GetData(MEMORYDEVICE_CUDA);

  const Vector2i leafSize(imgSize.x * imgSize.y, nbTrees);
  leaf_indices->ChangeDims(leafSize);
  int* leafData = leaf_indices->GetData(MEMORYDEVICE_CUDA);

  const dim3 blockSize(32, 32);
  const dim3 gridSize((imgSize.x + blockSize.x - 1) / blockSize.x,
      (imgSize.y + blockSize.y - 1) / blockSize.y);

  ck_evaluate_forest<<<gridSize,blockSize>>>(forestTexture, nbTrees, featureData, imgSize, leafData, leafSize);
  hipDeviceSynchronize();
}
}
