#include "hip/hip_runtime.h"
/**
 * spaint: ExampleReservoirs_CUDA.cu
 * Copyright (c) Torr Vision Group, University of Oxford, 2016. All rights reserved.
 */

#include "randomforest/cuda/ExampleReservoirs_CUDA.h"
#include "randomforest/cuda/ExampleReservoirs_CUDA.tcu"

namespace spaint
{

__global__ void ck_init_random_states(CUDARNG *randomStates, uint32_t nbStates,
    uint32_t seed)
{
  int idx = threadIdx.x + blockIdx.x * blockDim.x;

  if (idx >= nbStates)
    return;

  randomStates[idx].reset(seed, idx);
}

template<>
_CPU_AND_GPU_CODE_
inline PositionColourExample make_example_from_feature<PositionColourExample,
    RGBDPatchFeature>(const RGBDPatchFeature &feature)
{
  PositionColourExample res;
  res.position = feature.position.toVector3();
  res.colour = feature.colour;

  return res;
}

template class ExampleReservoirs_CUDA<PositionColourExample, RGBDPatchFeature,
    LeafIndices> ;
}
