#include "hip/hip_runtime.h"
/**
 * spaint: DepthVisualiser_CUDA.cu
 */

#include "visualisers/cuda/DepthVisualiser_CUDA.h"

#include "visualisers/shared/DepthVisualiser_Shared.h"

namespace spaint {

//#################### CUDA KERNELS ####################

__global__ void ck_render_depth(float *outRendering, Vector3f cameraPosition, Vector3f cameraLookVector, const Vector4f *ptsRay, Vector2i imgSize, float voxelSize, DepthVisualiser::DepthType depthType)
{
  int x = blockIdx.x * blockDim.x + threadIdx.x, y = blockIdx.y * blockDim.y + threadIdx.y;
  if(x >= imgSize.x || y >= imgSize.y) return;

  int locId = y * imgSize.x + x;
  Vector4f ptRay = ptsRay[locId];
  shade_pixel_depth(outRendering[locId], cameraPosition, cameraLookVector, ptRay.toVector3() * voxelSize, voxelSize, ptRay.w > 0, depthType);
}

//#################### PUBLIC MEMBER FUNCTIONS ####################

void DepthVisualiser_CUDA::render_depth(DepthType depthType, const Vector3f& cameraPosition, const Vector3f& cameraLookVector, const ITMLib::Objects::ITMRenderState *renderState,
                                        float voxelSize, const ITMFloatImage_Ptr& outputImage) const
{
  Vector2i imgSize = outputImage->noDims;

  // Shade all the pixels in the image.
  dim3 cudaBlockSize(8, 8);
  dim3 gridSize((int)ceil((float)imgSize.x / (float)cudaBlockSize.x), (int)ceil((float)imgSize.y / (float)cudaBlockSize.y));
  ck_render_depth<<<gridSize,cudaBlockSize>>>(
    outputImage->GetData(MEMORYDEVICE_CUDA),
    cameraPosition,
    cameraLookVector,
    renderState->raycastResult->GetData(MEMORYDEVICE_CUDA),
    imgSize,
    voxelSize,
    depthType
  );
}

}

