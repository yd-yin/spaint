#include "hip/hip_runtime.h"
/**
 * spaint: DepthVisualiser_CUDA.cu
 */

#include "visualisers/cuda/DepthVisualiser_CUDA.h"

#include "visualisers/shared/DepthVisualiser_Shared.h"

namespace spaint {

//#################### CUDA KERNELS ####################

__global__ void ck_render_depth(float *outRendering, Vector3f cameraPosition, Vector3f cameraLookVector, const Vector4f *ptsRay, Vector2i imgSize, float voxelSize, DepthVisualiser::DepthType depthType)
{
  int x = blockIdx.x * blockDim.x + threadIdx.x, y = blockIdx.y * blockDim.y + threadIdx.y;
  if(x >= imgSize.x || y >= imgSize.y) return;

  int locId = y * imgSize.x + x;
  Vector4f ptRay = ptsRay[locId];
  shade_pixel_depth(outRendering[locId], cameraPosition, cameraLookVector, ptRay.toVector3() * voxelSize, voxelSize, ptRay.w > 0, depthType);
}

//#################### PUBLIC MEMBER FUNCTIONS ####################

void DepthVisualiser_CUDA::render_depth(ITMFloatImage *outputImage, const ITMLib::Objects::ITMRenderState *renderState, Vector3f cameraPosition, Vector3f cameraLookVector, float voxelSize, DepthType depthType) const
{
  Vector2i imgSize = outputImage->noDims;

  // Shade all the pixels in the image.
  dim3 cudaBlockSize(8, 8);
  dim3 gridSize((int)ceil((float)imgSize.x / (float)cudaBlockSize.x), (int)ceil((float)imgSize.y / (float)cudaBlockSize.y));
  ck_render_depth<<<gridSize,cudaBlockSize>>>(
    outputImage->GetData(MEMORYDEVICE_CUDA),
    cameraPosition,
    cameraLookVector,
    renderState->raycastResult->GetData(MEMORYDEVICE_CUDA),
    imgSize,
    voxelSize,
    depthType
  );
}

}

