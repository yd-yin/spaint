#include "hip/hip_runtime.h"
/**
 * spaint: SemanticVisualiser_CUDA.cu
 */

#include "visualisers/cuda/SemanticVisualiser_CUDA.h"

#include "visualisers/shared/SemanticVisualiser_Shared.h"

namespace spaint {

//#################### CUDA KERNELS ####################

__global__ void ck_render_semantic(Vector4u *outRendering, const Vector4f *ptsRay, const SpaintVoxel *voxelData, const ITMVoxelIndex::IndexData *voxelIndex,
                                   Vector2i imgSize, Vector3u *labelColours, Vector3f viewerPos, Vector3f lightPos, bool usePhong)
{
  int x = blockIdx.x * blockDim.x + threadIdx.x, y = blockIdx.y * blockDim.y + threadIdx.y;
  if (x >= imgSize.x || y >= imgSize.y) return;

  int locId = y * imgSize.x + x;
  Vector4f ptRay = ptsRay[locId];
  shade_pixel_semantic(outRendering[locId], ptRay.toVector3(), ptRay.w > 0, voxelData, voxelIndex, labelColours, viewerPos, lightPos, usePhong);
}

//#################### PUBLIC MEMBER FUNCTIONS ####################

void SemanticVisualiser_CUDA::render(const ITMLib::Objects::ITMScene<SpaintVoxel,ITMVoxelIndex> *scene, const ITMLib::Objects::ITMPose *pose,
                                     const ITMLib::Objects::ITMIntrinsics *intrinsics, const ITMLib::Objects::ITMRenderState *renderState,
                                     const LabelManager *labelManager, bool usePhong, ITMUChar4Image *outputImage) const
{
  // Copy the label colours into a memory block.
  const std::vector<Vector3u>& labelColours = labelManager->get_label_colours();
  ORUtils::MemoryBlock<Vector3u> labelColoursMB(static_cast<int>(labelColours.size()), true, true);
  Vector3u *labelColoursData = labelColoursMB.GetData(MEMORYDEVICE_CPU);
  for(size_t i = 0, size = labelColours.size(); i < size; ++i)
  {
    labelColoursData[i] = labelColours[i];
  }
  labelColoursMB.UpdateDeviceFromHost();

  // Calculate the light and viewer positions in voxel coordinates (the same coordinate space as the raycast results).
  const float voxelSize = scene->sceneParams->voxelSize;
  Vector3f lightPos = Vector3f(0.0f, -10.0f, -10.0f) / voxelSize;
  Vector3f viewerPos = Vector3f(pose->GetInvM().getColumn(3)) / voxelSize;

  // Shade all of the pixels in the image.
  Vector2i imgSize = outputImage->noDims;

  dim3 cudaBlockSize(8, 8);
  dim3 gridSize((int)ceil((float)imgSize.x / (float)cudaBlockSize.x), (int)ceil((float)imgSize.y / (float)cudaBlockSize.y));

  ck_render_semantic<<<gridSize,cudaBlockSize>>>(
    outputImage->GetData(MEMORYDEVICE_CUDA),
    renderState->raycastResult->GetData(MEMORYDEVICE_CUDA),
    scene->localVBA.GetVoxelBlocks(),
    scene->index.getIndexData(),
    imgSize,
    labelColoursMB.GetData(MEMORYDEVICE_CUDA),
    viewerPos,
    lightPos,
    usePhong
  );
}

}
