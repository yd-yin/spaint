#include "hip/hip_runtime.h"
/**
 * spaint: VoxelToCubeSelectionTransformer_CUDA.cu
 */

#include "selectiontransformers/cuda/VoxelToCubeSelectionTransformer_CUDA.h"

#include "selectiontransformers/shared/VoxelToCubeSelectionTransformer_Shared.h"

namespace spaint {

//#################### CUDA KERNELS ####################

__global__ void ck_transform_selection(int cubeSideLength, int cubeSize, int radius, const Vector3s *inputSelection, Vector3s *outputSelection, int outputVoxelCount)
{
  int tid = blockDim.x * blockIdx.x + threadIdx.x;
  if(tid < outputVoxelCount) write_voxel_to_output_selection(tid, cubeSideLength, cubeSize, radius, inputSelection, outputSelection);
}

//#################### CONSTRUCTORS ####################

VoxelToCubeSelectionTransformer_CUDA::VoxelToCubeSelectionTransformer_CUDA(int radius)
: VoxelToCubeSelectionTransformer(radius, ITMLibSettings::DEVICE_CUDA)
{}

//#################### PUBLIC MEMBER FUNCTIONS ####################

void VoxelToCubeSelectionTransformer_CUDA::transform_selection(const Selection& inputSelectionMB, Selection& outputSelectionMB) const
{
  int outputVoxelCount = static_cast<int>(outputSelectionMB.dataSize);

  int threadsPerBlock = 256;
  int numBlocks = (outputVoxelCount + threadsPerBlock - 1) / threadsPerBlock;

  ck_transform_selection<<<numBlocks,threadsPerBlock>>>(
    cube_side_length(),
    cube_size(),
    m_radius,
    inputSelectionMB.GetData(MEMORYDEVICE_CUDA),
    outputSelectionMB.GetData(MEMORYDEVICE_CUDA),
    outputVoxelCount
  );
}

}
