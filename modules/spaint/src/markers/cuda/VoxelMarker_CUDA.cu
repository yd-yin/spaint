#include "hip/hip_runtime.h"
/**
 * spaint: VoxelMarker_CUDA.cu
 */

#include "markers/cuda/VoxelMarker_CUDA.h"

#include "markers/shared/VoxelMarker_Shared.h"

namespace spaint {

//#################### CUDA KERNELS ####################

__global__ void ck_mark_voxels(const Vector3s *voxelLocations, unsigned char label, int voxelCount, unsigned char *oldVoxelLabels,
                               SpaintVoxel *voxelData, const ITMVoxelIndex::IndexData *voxelIndex)
{
  int tid = blockDim.x * blockIdx.x + threadIdx.x;
  if(tid < voxelCount) mark_voxel(voxelLocations[tid], label, oldVoxelLabels ? &oldVoxelLabels[tid] : NULL, voxelData, voxelIndex);
}

__global__ void ck_mark_voxels(const Vector3s *voxelLocations, const unsigned char *voxelLabels, int voxelCount, unsigned char *oldVoxelLabels,
                               SpaintVoxel *voxelData, const ITMVoxelIndex::IndexData *voxelIndex)
{
  int tid = blockDim.x * blockIdx.x + threadIdx.x;
  if(tid < voxelCount) mark_voxel(voxelLocations[tid], voxelLabels[tid], oldVoxelLabels ? &oldVoxelLabels[tid] : NULL, voxelData, voxelIndex);
}

//#################### PUBLIC MEMBER FUNCTIONS ####################

void VoxelMarker_CUDA::mark_voxels(const ORUtils::MemoryBlock<Vector3s>& voxelLocationsMB, unsigned char label,
                                   ITMLib::Objects::ITMScene<SpaintVoxel,ITMVoxelIndex> *scene,
                                   ORUtils::MemoryBlock<unsigned char> *oldVoxelLabelsMB) const
{
  int voxelCount = voxelLocationsMB.dataSize;

  int threadsPerBlock = 256;
  int numBlocks = (voxelCount + threadsPerBlock - 1) / threadsPerBlock;

  ck_mark_voxels<<<numBlocks,threadsPerBlock>>>(
    voxelLocationsMB.GetData(MEMORYDEVICE_CUDA),
    label,
    voxelCount,
    oldVoxelLabelsMB ? oldVoxelLabelsMB->GetData(MEMORYDEVICE_CUDA) : NULL,
    scene->localVBA.GetVoxelBlocks(),
    scene->index.getIndexData()
  );
}

void VoxelMarker_CUDA::mark_voxels(const ORUtils::MemoryBlock<Vector3s>& voxelLocationsMB,
                                   const ORUtils::MemoryBlock<unsigned char>& voxelLabelsMB,
                                   ITMLib::Objects::ITMScene<SpaintVoxel,ITMVoxelIndex> *scene,
                                   ORUtils::MemoryBlock<unsigned char> *oldVoxelLabelsMB) const
{
  int voxelCount = voxelLocationsMB.dataSize;

  int threadsPerBlock = 256;
  int numBlocks = (voxelCount + threadsPerBlock - 1) / threadsPerBlock;

  ck_mark_voxels<<<numBlocks,threadsPerBlock>>>(
    voxelLocationsMB.GetData(MEMORYDEVICE_CUDA),
    voxelLabelsMB.GetData(MEMORYDEVICE_CUDA),
    voxelCount,
    oldVoxelLabelsMB ? oldVoxelLabelsMB->GetData(MEMORYDEVICE_CUDA) : NULL,
    scene->localVBA.GetVoxelBlocks(),
    scene->index.getIndexData()
  );
}

}
