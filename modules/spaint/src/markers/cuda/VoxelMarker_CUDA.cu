#include "hip/hip_runtime.h"
/**
 * spaint: VoxelMarker_CUDA.cu
 */

#include "markers/cuda/VoxelMarker_CUDA.h"

#include "markers/shared/VoxelMarker_Shared.h"

namespace spaint {

//#################### CUDA KERNELS ####################

__global__ void ck_clear_labels(SpaintVoxel *voxels, int voxelCount)
{
  int tid = blockDim.x * blockIdx.x + threadIdx.x;
  if(tid < voxelCount) voxels[tid].packedLabel = SpaintVoxel::PackedLabel();
}

__global__ void ck_mark_voxels(const Vector3s *voxelLocations, SpaintVoxel::PackedLabel label, int voxelCount, SpaintVoxel::PackedLabel *oldVoxelLabels,
                               SpaintVoxel *voxelData, const ITMVoxelIndex::IndexData *voxelIndex)
{
  int tid = blockDim.x * blockIdx.x + threadIdx.x;
  if(tid < voxelCount) mark_voxel(voxelLocations[tid], label, oldVoxelLabels ? &oldVoxelLabels[tid] : NULL, voxelData, voxelIndex);
}

__global__ void ck_mark_voxels(const Vector3s *voxelLocations, const SpaintVoxel::PackedLabel *voxelLabels, int voxelCount, SpaintVoxel::PackedLabel *oldVoxelLabels,
                               SpaintVoxel *voxelData, const ITMVoxelIndex::IndexData *voxelIndex)
{
  int tid = blockDim.x * blockIdx.x + threadIdx.x;
  if(tid < voxelCount) mark_voxel(voxelLocations[tid], voxelLabels[tid], oldVoxelLabels ? &oldVoxelLabels[tid] : NULL, voxelData, voxelIndex);
}

//#################### PUBLIC MEMBER FUNCTIONS ####################

void VoxelMarker_CUDA::clear_labels(SpaintVoxel *voxels, int voxelCount) const
{
  int threadsPerBlock = 256;
  int numBlocks = (voxelCount + threadsPerBlock - 1) / threadsPerBlock;

  ck_clear_labels<<<numBlocks,threadsPerBlock>>>(voxels, voxelCount);
}

void VoxelMarker_CUDA::mark_voxels(const ORUtils::MemoryBlock<Vector3s>& voxelLocationsMB, SpaintVoxel::PackedLabel label,
                                   ITMLib::Objects::ITMScene<SpaintVoxel,ITMVoxelIndex> *scene,
                                   ORUtils::MemoryBlock<SpaintVoxel::PackedLabel> *oldVoxelLabelsMB) const
{
  int voxelCount = static_cast<int>(voxelLocationsMB.dataSize);

  int threadsPerBlock = 256;
  int numBlocks = (voxelCount + threadsPerBlock - 1) / threadsPerBlock;

  ck_mark_voxels<<<numBlocks,threadsPerBlock>>>(
    voxelLocationsMB.GetData(MEMORYDEVICE_CUDA),
    label,
    voxelCount,
    oldVoxelLabelsMB ? oldVoxelLabelsMB->GetData(MEMORYDEVICE_CUDA) : NULL,
    scene->localVBA.GetVoxelBlocks(),
    scene->index.getIndexData()
  );
}

void VoxelMarker_CUDA::mark_voxels(const ORUtils::MemoryBlock<Vector3s>& voxelLocationsMB,
                                   const ORUtils::MemoryBlock<SpaintVoxel::PackedLabel>& voxelLabelsMB,
                                   ITMLib::Objects::ITMScene<SpaintVoxel,ITMVoxelIndex> *scene,
                                   ORUtils::MemoryBlock<SpaintVoxel::PackedLabel> *oldVoxelLabelsMB) const
{
  int voxelCount = static_cast<int>(voxelLocationsMB.dataSize);

  int threadsPerBlock = 256;
  int numBlocks = (voxelCount + threadsPerBlock - 1) / threadsPerBlock;

  ck_mark_voxels<<<numBlocks,threadsPerBlock>>>(
    voxelLocationsMB.GetData(MEMORYDEVICE_CUDA),
    voxelLabelsMB.GetData(MEMORYDEVICE_CUDA),
    voxelCount,
    oldVoxelLabelsMB ? oldVoxelLabelsMB->GetData(MEMORYDEVICE_CUDA) : NULL,
    scene->localVBA.GetVoxelBlocks(),
    scene->index.getIndexData()
  );
}

}
