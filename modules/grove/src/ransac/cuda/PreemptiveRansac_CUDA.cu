#include "hip/hip_runtime.h"
/**
 * grove: PreemptiveRansac_CUDA.cu
 * Copyright (c) Torr Vision Group, University of Oxford, 2017. All rights reserved.
 */

#include "ransac/cuda/PreemptiveRansac_CUDA.h"
using namespace tvgutil;

#ifdef _MSC_VER
  // Suppress some VC++ warnings that are produced when including the Thrust headers.
  #pragma warning(disable:4244 4267)
#endif

#include <thrust/device_ptr.h>
#include <thrust/sort.h>

#ifdef _MSC_VER
  // Reenable the suppressed warnings for the rest of the translation unit.
  #pragma warning(default:4244 4267)
#endif

#include <orx/base/MemoryBlockFactory.h>
using namespace orx;

#include "ransac/shared/PreemptiveRansac_Shared.h"

namespace grove {

//#################### CUDA KERNELS ####################

__global__ void ck_compute_energies(const Keypoint3DColour *keypoints, const ScorePrediction *predictions, const int *inlierRasterIndices,
                                    uint32_t nbInliers, PoseCandidate *poseCandidates, int nbCandidates)
{
  const int tid = threadIdx.x;
  const int threadsPerBlock = blockDim.x;
  const int candidateIdx = blockIdx.x;

  if(candidateIdx >= nbCandidates)
  {
    // The candidate has been culled, so early out. Note that since we are using each thread block to
    // compute the energy for a single candidate, the entire block will return in this case, so the
    // __syncthreads() call later in the kernel is safe.
    return;
  }

  PoseCandidate& currentCandidate = poseCandidates[candidateIdx];

  // For each thread in the block, first compute the sum of the energies for a strided subset of the inliers.
  // In particular, thread tid in the block computes the sum of the energies for the inliers with array indices
  // tid + k * threadsPerBlock.
  float energySum = compute_energy_sum_for_inlier_subset(
    currentCandidate.cameraPose, keypoints, predictions, inlierRasterIndices, nbInliers, tid, threadsPerBlock
  );

  // Then, add up the sums computed by the individual threads to compute the overall energy for the candidate.
  // To do this, we perform an efficient, shuffle-based reduction as described in the following blog post:
  // https://devblogs.nvidia.com/parallelforall/faster-parallel-reductions-kepler

  // Step 1: Sum the energies in each warp using downward shuffling, storing the result in the energySum variable of the first thread in the warp.
  for(int offset = warpSize / 2; offset > 0; offset /= 2)
#if defined(__CUDACC_VER_MAJOR__) && (__CUDACC_VER_MAJOR__ >= 9)
    energySum += __shfl_down_sync(0xFFFFFFFF, energySum, offset);
#else
    energySum += __shfl_down(energySum, offset);
#endif

  // Step 2: If this is the first thread in the warp, add the energy sum for the warp to the candidate's energy.
  if((threadIdx.x & (warpSize - 1)) == 0) atomicAdd(&currentCandidate.energy, energySum);

  // Step 3: Wait for all of the atomic adds to finish.
  __syncthreads();

  // Step 4: If this is the first thread in the entire block, compute the final energy for the candidate by dividing by the number of inliers.
  if(tid == 0) currentCandidate.energy = currentCandidate.energy / static_cast<float>(nbInliers);
}

template <typename RNG>
__global__ void ck_generate_pose_candidates(const Keypoint3DColour *keypoints, const ScorePrediction *predictions,
                                            const Vector2i imgSize, RNG *rngs, PoseCandidate *poseCandidates, int *nbPoseCandidates,
                                            uint32_t maxCandidateGenerationIterations, uint32_t maxPoseCandidates,
                                            bool useAllModesPerLeafInPoseHypothesisGeneration, bool checkMinDistanceBetweenSampledModes,
                                            float minDistanceBetweenSampledModes, bool checkRigidTransformationConstraint,
                                            float translationErrorMaxForCorrectPose)
{
  const int candidateIdx = blockIdx.x * blockDim.x + threadIdx.x;
  if(candidateIdx >= maxPoseCandidates) return;

  // Try to generate a valid pose candidate.
  PoseCandidate candidate;
  bool valid = generate_pose_candidate(
    keypoints, predictions, imgSize, rngs[candidateIdx], candidate, maxCandidateGenerationIterations, useAllModesPerLeafInPoseHypothesisGeneration,
    checkMinDistanceBetweenSampledModes, minDistanceBetweenSampledModes, checkRigidTransformationConstraint, translationErrorMaxForCorrectPose
  );

  // If we succeed, grab a unique index in the output array and store the candidate into the corresponding array element.
  if(valid)
  {
    const int finalCandidateIdx = atomicAdd(nbPoseCandidates, 1);
    poseCandidates[finalCandidateIdx] = candidate;
  }
}

__global__ void ck_prepare_inliers_for_optimisation(const Keypoint3DColour *keypoints, const ScorePrediction *predictions, const int *inlierIndices, int nbInliers,
                                                    const PoseCandidate *poseCandidates, int nbPoseCandidates, float inlierThreshold, Vector4f *inlierCameraPoints,
                                                    Keypoint3DColourCluster *inlierModes)
{
  const int candidateIdx = blockIdx.y;
  const int inlierIdx = blockIdx.x * blockDim.x + threadIdx.x;

  if(candidateIdx < nbPoseCandidates && inlierIdx < nbInliers)
  {
    prepare_inlier_for_optimisation(
      candidateIdx, inlierIdx, keypoints, predictions, inlierIndices, nbInliers, poseCandidates, inlierThreshold, inlierCameraPoints, inlierModes
    );
  }
}

__global__ void ck_reset_candidate_energies(PoseCandidate *poseCandidates, int nbPoseCandidates)
{
  const int candidateIdx = blockIdx.x * blockDim.x + threadIdx.x;
  if(candidateIdx < nbPoseCandidates)
  {
    poseCandidates[candidateIdx].energy = 0.0f;
  }
}

template <bool useMask, typename RNG>
__global__ void ck_sample_inliers(const Keypoint3DColour *keypoints, const ScorePrediction *predictions, const Vector2i imgSize, RNG *rngs,
                                  int *inlierRasterIndices, int *nbInliers, uint32_t ransacInliersPerIteration, int *inliersMask = NULL)
{
  const uint32_t sampleIdx = blockIdx.x * blockDim.x + threadIdx.x;
  if(sampleIdx < ransacInliersPerIteration)
  {
    // Try to sample the raster index of a valid keypoint which prediction has at least one modal cluster, using the mask if necessary.
    const int rasterIdx = sample_inlier<useMask>(keypoints, predictions, imgSize, rngs[sampleIdx], inliersMask);

    // If we succeed, grab a unique index in the output array and store the inlier raster index into the corresponding array element.
    if(rasterIdx >= 0)
    {
      const int arrayIdx = atomicAdd(nbInliers, 1);
      inlierRasterIndices[arrayIdx] = rasterIdx;
    }
  }
}

//#################### CONSTRUCTORS ####################

PreemptiveRansac_CUDA::PreemptiveRansac_CUDA(const SettingsContainer_CPtr& settings, const std::string& settingsNamespace)
: PreemptiveRansac(settings, settingsNamespace)
{
  MemoryBlockFactory& mbf = MemoryBlockFactory::instance();

  // Allocate memory blocks.
  m_nbInliers_device = mbf.make_block<int>(1);        // Size 1, just to store a value that can be accessed from the GPU.
  m_nbPoseCandidates_device = mbf.make_block<int>(1); // As above.
  m_rngs = mbf.make_block<CUDARNG>(m_maxPoseCandidates);

  // Default random seed.
  m_rngSeed = 42;

  // Reset RNGs.
  init_random();
}

//#################### PROTECTED MEMBER FUNCTIONS ####################

void PreemptiveRansac_CUDA::compute_energies_and_sort()
{
  const int *inlierRasterIndices = m_inlierRasterIndicesBlock->GetData(MEMORYDEVICE_CUDA);
  const Keypoint3DColour *keypoints = m_keypointsImage->GetData(MEMORYDEVICE_CUDA);
  const uint32_t nbInliers = static_cast<uint32_t>(m_inlierRasterIndicesBlock->dataSize); // The number of currently sampled inlier points (used to compute the energy).
  const int nbPoseCandidates = static_cast<int>(m_poseCandidates->dataSize);              // The number of currently "valid" pose candidates.
  PoseCandidate *poseCandidates = m_poseCandidates->GetData(MEMORYDEVICE_CUDA);           // The raster indices of the current sampled inlier points.
  const ScorePrediction *predictions = m_predictionsImage->GetData(MEMORYDEVICE_CUDA);

  // Reset the energies for all pose candidates.
  {
    dim3 blockSize(256);
    dim3 gridSize((nbPoseCandidates + blockSize.x - 1) / blockSize.x);
    ck_reset_candidate_energies<<<gridSize,blockSize>>>(poseCandidates, nbPoseCandidates);
    ORcudaKernelCheck;
  }

  // Compute the energies for all pose candidates.
  {
    // Launch one block per candidate (in this way, many blocks will exit immediately in the later stages of P-RANSAC).
    dim3 blockSize(128); // Threads to compute the energy for each candidate.
    dim3 gridSize(nbPoseCandidates);
    ck_compute_energies<<<gridSize,blockSize>>>(keypoints, predictions, inlierRasterIndices, nbInliers, poseCandidates, nbPoseCandidates);
    ORcudaKernelCheck;
  }

  // Sort the candidates into non-decreasing order of energy.
  thrust::device_ptr<PoseCandidate> candidatesStart(poseCandidates);
  thrust::device_ptr<PoseCandidate> candidatesEnd(poseCandidates + nbPoseCandidates);
  thrust::sort(candidatesStart, candidatesEnd);
}

void PreemptiveRansac_CUDA::generate_pose_candidates()
{
  const Vector2i imgSize = m_keypointsImage->noDims;
  const Keypoint3DColour *keypoints = m_keypointsImage->GetData(MEMORYDEVICE_CUDA);
  PoseCandidate *poseCandidates = m_poseCandidates->GetData(MEMORYDEVICE_CUDA);
  const ScorePrediction *predictions = m_predictionsImage->GetData(MEMORYDEVICE_CUDA);
  CUDARNG *rngs = m_rngs->GetData(MEMORYDEVICE_CUDA);

  // Reset the number of pose candidates (we do this on the device only at this stage, and update the corresponding host value once we are done generating).
  int *nbPoseCandidates_device = m_nbPoseCandidates_device->GetData(MEMORYDEVICE_CUDA);
  ORcudaSafeCall(hipMemsetAsync(nbPoseCandidates_device, 0, sizeof(int)));

  // Generate at most m_maxPoseCandidates new pose candidates.
  dim3 blockSize(32);
  dim3 gridSize((m_maxPoseCandidates + blockSize.x - 1) / blockSize.x);

  ck_generate_pose_candidates<<<gridSize,blockSize>>>(
    keypoints, predictions, imgSize, rngs, poseCandidates, nbPoseCandidates_device, m_maxCandidateGenerationIterations,
    m_maxPoseCandidates, m_useAllModesPerLeafInPoseHypothesisGeneration, m_checkMinDistanceBetweenSampledModes,
    m_minSquaredDistanceBetweenSampledModes, m_checkRigidTransformationConstraint, m_maxTranslationErrorForCorrectPose
  );
  ORcudaKernelCheck;

  // Copy all relevant data back across to the host for use by the Kabsch algorithm.
  m_poseCandidates->dataSize = m_nbPoseCandidates_device->GetElement(0, MEMORYDEVICE_CUDA);
  m_poseCandidates->UpdateHostFromDevice();

  // Run Kabsch on all the generated candidates to estimate the rigid transformations.
  compute_candidate_poses_kabsch();

  // Copy the computed rigid transformations back across to the device.
  m_poseCandidates->UpdateDeviceFromHost();
}

void PreemptiveRansac_CUDA::prepare_inliers_for_optimisation()
{
  Vector4f *inlierCameraPoints = m_poseOptimisationCameraPoints->GetData(MEMORYDEVICE_CUDA);
  Keypoint3DColourCluster *inlierModes = m_poseOptimisationPredictedModes->GetData(MEMORYDEVICE_CUDA);
  const int *inlierRasterIndices = m_inlierRasterIndicesBlock->GetData(MEMORYDEVICE_CUDA);
  const Keypoint3DColour *keypoints = m_keypointsImage->GetData(MEMORYDEVICE_CUDA);
  const uint32_t nbInliers = static_cast<uint32_t>(m_inlierRasterIndicesBlock->dataSize);
  const int nbPoseCandidates = static_cast<int>(m_poseCandidates->dataSize);
  const PoseCandidate *poseCandidates = m_poseCandidates->GetData(MEMORYDEVICE_CUDA);
  const ScorePrediction *predictions = m_predictionsImage->GetData(MEMORYDEVICE_CUDA);

  dim3 blockSize(256);
  dim3 gridSize((nbInliers + blockSize.x - 1) / blockSize.x, nbPoseCandidates);

  ck_prepare_inliers_for_optimisation<<<gridSize, blockSize>>>(
    keypoints, predictions, inlierRasterIndices, nbInliers, poseCandidates, nbPoseCandidates,
    m_poseOptimisationInlierThreshold, inlierCameraPoints, inlierModes
  );
  ORcudaKernelCheck;

  // Compute and set the actual size of the buffers to avoid unnecessary copies.
  const size_t bufferSize = static_cast<size_t>(nbInliers * nbPoseCandidates);
  m_poseOptimisationCameraPoints->dataSize = bufferSize;
  m_poseOptimisationPredictedModes->dataSize = bufferSize;

  // Make the buffers available to the optimiser, which runs on the CPU.
  m_poseOptimisationCameraPoints->UpdateHostFromDevice();
  m_poseOptimisationPredictedModes->UpdateHostFromDevice();
}

void PreemptiveRansac_CUDA::reset_inliers(bool resetMask)
{
  PreemptiveRansac::reset_inliers(resetMask);
  ORcudaSafeCall(hipMemsetAsync(m_nbInliers_device->GetData(MEMORYDEVICE_CUDA), 0, sizeof(int)));
}

void PreemptiveRansac_CUDA::sample_inliers(bool useMask)
{
  const Vector2i imgSize = m_keypointsImage->noDims;
  int *inlierRasterIndices = m_inlierRasterIndicesBlock->GetData(MEMORYDEVICE_CUDA);
  int *inliersMask = m_inliersMaskImage->GetData(MEMORYDEVICE_CUDA);
  const Keypoint3DColour *keypoints = m_keypointsImage->GetData(MEMORYDEVICE_CUDA);
  int *nbInliers_device = m_nbInliers_device->GetData(MEMORYDEVICE_CUDA);
  const ScorePrediction *predictions = m_predictionsImage->GetData(MEMORYDEVICE_CUDA);
  CUDARNG *rngs = m_rngs->GetData(MEMORYDEVICE_CUDA);

  dim3 blockSize(128);
  dim3 gridSize((m_ransacInliersPerIteration + blockSize.x - 1) / blockSize.x);

  if(useMask)
  {
    ck_sample_inliers<true><<<gridSize,blockSize>>>(
      keypoints, predictions, imgSize, rngs, inlierRasterIndices, nbInliers_device, m_ransacInliersPerIteration, inliersMask
    );
    ORcudaKernelCheck;
  }
  else
  {
    ck_sample_inliers<false><<<gridSize,blockSize>>>(
      keypoints, predictions, imgSize, rngs, inlierRasterIndices, nbInliers_device, m_ransacInliersPerIteration
    );
    ORcudaKernelCheck;
  }

  // Update the host's record of the number of inliers.
  m_inlierRasterIndicesBlock->dataSize = static_cast<size_t>(m_nbInliers_device->GetElement(0, MEMORYDEVICE_CUDA));
}

void PreemptiveRansac_CUDA::update_candidate_poses()
{
  // Copy the pose candidates across to the CPU so that they can be optimised.
  m_poseCandidates->UpdateHostFromDevice();

  // Call the base class implementation to optimise the poses.
  PreemptiveRansac::update_candidate_poses();

  // Copy the optimised pose candidates back across to the GPU.
  m_poseCandidates->UpdateDeviceFromHost();
}

//#################### PRIVATE MEMBER FUNCTIONS ####################

void PreemptiveRansac_CUDA::init_random()
{
  CUDARNG *rngs = m_rngs->GetData(MEMORYDEVICE_CUDA);

  // Initialize random states
  dim3 blockSize(256);
  dim3 gridSize((m_maxPoseCandidates + blockSize.x - 1) / blockSize.x);

  ck_reinit_rngs<<<gridSize, blockSize>>>(rngs, m_maxPoseCandidates, m_rngSeed);
  ORcudaKernelCheck;
}

void PreemptiveRansac_CUDA::update_host_pose_candidates() const
{
  m_poseCandidates->UpdateHostFromDevice();
}

}
