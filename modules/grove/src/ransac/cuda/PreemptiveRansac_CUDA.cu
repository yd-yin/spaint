#include "hip/hip_runtime.h"
/**
 * grove: PreemptiveRansac_CUDA.cu
 * Copyright (c) Torr Vision Group, University of Oxford, 2017. All rights reserved.
 */

#include "ransac/cuda/PreemptiveRansac_CUDA.h"

#ifdef _MSC_VER
  // Suppress some VC++ warnings that are produced when including the Thrust headers.
  #pragma warning(disable:4244 4267)
#endif

#include <thrust/device_ptr.h>
#include <thrust/sort.h>

#ifdef _MSC_VER
  // Reenable the suppressed warnings for the rest of the translation unit.
  #pragma warning(default:4244 4267)
#endif

#include <itmx/base/MemoryBlockFactory.h>

#include "ransac/shared/PreemptiveRansac_Shared.h"

using namespace itmx;
using namespace tvgutil;

namespace grove {

//#################### CUDA KERNELS ####################

__global__ void ck_preemptive_ransac_compute_energies(const Keypoint3DColour *keypoints,
                                                      const ScorePrediction *predictions,
                                                      const int *inlierRasterIndices,
                                                      uint32_t nbInliers,
                                                      PoseCandidate *poseCandidates,
                                                      int nbCandidates)
{
  const int tId = threadIdx.x;
  const int threadsPerBlock = blockDim.x;
  const int candidateIdx = blockIdx.x;

  if(candidateIdx >= nbCandidates)
  {
    // Candidate has been culled.
    // Since the entire block returns, this does not cause troubles with the following __syncthreads()
    return;
  }

  PoseCandidate& currentCandidate = poseCandidates[candidateIdx];

  // Compute the energy for a strided subset of inliers.
  float localEnergy = preemptive_ransac_compute_candidate_energy(
      currentCandidate.cameraPose, keypoints, predictions, inlierRasterIndices, nbInliers, tId, threadsPerBlock);

  // The reduction is performed as in the following blog post:
  // https://devblogs.nvidia.com/parallelforall/faster-parallel-reductions-kepler

  // Reduce by shuffling down the local energies (localEnergy for thread 0 in the warp contains the sum for the warp).
  for(int offset = warpSize / 2; offset > 0; offset /= 2) localEnergy += __shfl_down(localEnergy, offset);

  // Thread 0 of each warp atomically updates the final energy.
  if((threadIdx.x & (warpSize - 1)) == 0) atomicAdd(&currentCandidate.energy, localEnergy);

  __syncthreads(); // Wait for all threads in the block

  // tId 0 computes the final energy
  if(tId == 0) currentCandidate.energy = currentCandidate.energy / static_cast<float>(nbInliers);
}

template <typename RNG>
__global__ void ck_preemptive_ransac_generate_pose_candidates(const Keypoint3DColour *keypoints,
                                                              const ScorePrediction *predictions,
                                                              const Vector2i imgSize,
                                                              RNG *randomGenerators,
                                                              PoseCandidate *poseCandidates,
                                                              int *nbPoseCandidates,
                                                              uint32_t maxCandidateGenerationIterations,
                                                              uint32_t maxPoseCandidates,
                                                              bool useAllModesPerLeafInPoseHypothesisGeneration,
                                                              bool checkMinDistanceBetweenSampledModes,
                                                              float minDistanceBetweenSampledModes,
                                                              bool checkRigidTransformationConstraint,
                                                              float translationErrorMaxForCorrectPose)
{
  const int candidateIdx = blockIdx.x * blockDim.x + threadIdx.x;

  if(candidateIdx >= maxPoseCandidates) return;

  // Try to generate a candidate in a local variable.
  PoseCandidate candidate;

  bool valid = preemptive_ransac_generate_candidate(keypoints,
                                                    predictions,
                                                    imgSize,
                                                    randomGenerators[candidateIdx],
                                                    candidate,
                                                    maxCandidateGenerationIterations,
                                                    useAllModesPerLeafInPoseHypothesisGeneration,
                                                    checkMinDistanceBetweenSampledModes,
                                                    minDistanceBetweenSampledModes,
                                                    checkRigidTransformationConstraint,
                                                    translationErrorMaxForCorrectPose);

  // If we succeeded, grab an unique index and store the candidate in the array.
  if(valid)
  {
    const int finalCandidateIdx = atomicAdd(nbPoseCandidates, 1);
    poseCandidates[finalCandidateIdx] = candidate;
  }
}

__global__ void ck_preemptive_ransac_prepare_inliers_for_optimisation(const Keypoint3DColour *keypoints,
                                                                      const ScorePrediction *predictions,
                                                                      const int *inlierIndices,
                                                                      int nbInliers,
                                                                      const PoseCandidate *poseCandidates,
                                                                      int nbPoseCandidates,
                                                                      Vector4f *inlierCameraPoints,
                                                                      Keypoint3DColourCluster *inlierModes,
                                                                      float inlierThreshold)
{
  const int candidateIdx = blockIdx.y;
  const int inlierIdx = blockIdx.x * blockDim.x + threadIdx.x;

  if(candidateIdx >= nbPoseCandidates || inlierIdx >= nbInliers) return;

  preemptive_ransac_prepare_inliers_for_optimisation(keypoints,
                                                     predictions,
                                                     inlierIndices,
                                                     nbInliers,
                                                     poseCandidates,
                                                     inlierCameraPoints,
                                                     inlierModes,
                                                     inlierThreshold,
                                                     candidateIdx,
                                                     inlierIdx);
}

__global__ void ck_preemptive_ransac_reset_candidate_energies(PoseCandidate *poseCandidates, int nbPoseCandidates)
{
  const int candidateIdx = blockIdx.x * blockDim.x + threadIdx.x;

  if(candidateIdx >= nbPoseCandidates)
  {
    return;
  }

  poseCandidates[candidateIdx].energy = 0.f;
}

template <bool useMask, typename RNG>
__global__ void ck_preemptive_ransac_sample_inliers(const Keypoint3DColour *keypointsData,
                                                    const ScorePrediction *predictionsData,
                                                    const Vector2i imgSize,
                                                    RNG *randomGenerators,
                                                    int *inlierIndices,
                                                    int *inlierCount,
                                                    uint32_t nbMaxSamples,
                                                    int *inlierMaskData = NULL)
{
  const uint32_t sampleIdx = blockIdx.x * blockDim.x + threadIdx.x;

  if(sampleIdx >= nbMaxSamples) return;

  // Try to sample the raster index of a valid keypoint which prediction has at least one modal cluster, using the mask
  // if necessary.
  const int sampledLinearIdx = preemptive_ransac_sample_inlier<useMask>(
      keypointsData, predictionsData, imgSize, randomGenerators[sampleIdx], inlierMaskData);

  // If the sampling succeeded grab a global index and store the keypoint index.
  if(sampledLinearIdx >= 0)
  {
    const int outIdx = atomicAdd(inlierCount, 1);
    inlierIndices[outIdx] = sampledLinearIdx;
  }
}

//#################### CONSTRUCTORS ####################

PreemptiveRansac_CUDA::PreemptiveRansac_CUDA(const SettingsContainer_CPtr& settings)
: PreemptiveRansac(settings)
{
  MemoryBlockFactory& mbf = MemoryBlockFactory::instance();

  // Allocate memory blocks.
  m_nbPoseCandidates_device =
      mbf.make_block<int>(1); // Size 1, just to store a value that can be accessed from the GPU.
  m_nbSampledInliers_device = mbf.make_block<int>(1); // As above.
  m_randomGenerators = mbf.make_block<CUDARNG>(m_maxPoseCandidates);

  // Default random seed.
  m_rngSeed = 42;

  // Reset RNGs.
  init_random();
}

//#################### PROTECTED VIRTUAL MEMBER FUNCTIONS ####################

void PreemptiveRansac_CUDA::compute_energies_and_sort()
{
  // Number of currently sampled inlier points, used to compute the energy.
  const size_t nbInliers = m_inliersIndicesBlock->dataSize;
  // Number of currently "valid" pose candidates.
  const size_t nbPoseCandidates = m_poseCandidates->dataSize;

  const Keypoint3DColour *keypoints = m_keypointsImage->GetData(MEMORYDEVICE_CUDA);
  const ScorePrediction *predictions = m_predictionsImage->GetData(MEMORYDEVICE_CUDA);
  // Indices of the sampled inliers.
  const int *inlierRasterIndices = m_inliersIndicesBlock->GetData(MEMORYDEVICE_CUDA);

  PoseCandidate *poseCandidates = m_poseCandidates->GetData(MEMORYDEVICE_CUDA);

  // First, reset the energy values.
  {
    dim3 blockSize(256);
    dim3 gridSize((nbPoseCandidates + blockSize.x - 1) / blockSize.x);
    ck_preemptive_ransac_reset_candidate_energies<<<gridSize, blockSize>>>(poseCandidates, nbPoseCandidates);
    ORcudaKernelCheck;
  }

  // Then compute the energies.
  {
    // Launch one block per candidate (in this way many blocks will exit immediately in later stages of P-RANSAC).
    dim3 blockSize(128); // Threads to compute the energy for each candidate.
    dim3 gridSize(nbPoseCandidates);
    ck_preemptive_ransac_compute_energies<<<gridSize, blockSize>>>(
        keypoints, predictions, inlierRasterIndices, nbInliers, poseCandidates, nbPoseCandidates);
    ORcudaKernelCheck;
  }

  // Finally, sort candidates by ascending energy using operator <.
  thrust::device_ptr<PoseCandidate> candidatesStart(poseCandidates);
  thrust::device_ptr<PoseCandidate> candidatesEnd(poseCandidates + nbPoseCandidates);
  thrust::sort(candidatesStart, candidatesEnd);
}

void PreemptiveRansac_CUDA::generate_pose_candidates()
{
  const Vector2i imgSize = m_keypointsImage->noDims;
  const Keypoint3DColour *keypoints = m_keypointsImage->GetData(MEMORYDEVICE_CUDA);
  const ScorePrediction *predictions = m_predictionsImage->GetData(MEMORYDEVICE_CUDA);

  CUDARNG *randomGenerators = m_randomGenerators->GetData(MEMORYDEVICE_CUDA);
  PoseCandidate *poseCandidates = m_poseCandidates->GetData(MEMORYDEVICE_CUDA);
  int *nbPoseCandidates_device = m_nbPoseCandidates_device->GetData(MEMORYDEVICE_CUDA);

  dim3 blockSize(32);
  dim3 gridSize((m_maxPoseCandidates + blockSize.x - 1) / blockSize.x);

  // Reset number of candidates (device only, the host number will be updated later, when we are done generating).
  ORcudaSafeCall(hipMemsetAsync(nbPoseCandidates_device, 0, sizeof(int)));

  ck_preemptive_ransac_generate_pose_candidates<<<gridSize, blockSize>>>(keypoints,
                                                                         predictions,
                                                                         imgSize,
                                                                         randomGenerators,
                                                                         poseCandidates,
                                                                         nbPoseCandidates_device,
                                                                         m_maxCandidateGenerationIterations,
                                                                         m_maxPoseCandidates,
                                                                         m_useAllModesPerLeafInPoseHypothesisGeneration,
                                                                         m_checkMinDistanceBetweenSampledModes,
                                                                         m_minSquaredDistanceBetweenSampledModes,
                                                                         m_checkRigidTransformationConstraint,
                                                                         m_maxTranslationErrorForCorrectPose);
  ORcudaKernelCheck;

  // Need to make the data available to the host (for Kabsch).
  m_poseCandidates->dataSize = m_nbPoseCandidates_device->GetElement(0, MEMORYDEVICE_CUDA);
  m_poseCandidates->UpdateHostFromDevice();

  // Now perform kabsch on all candidates.
  compute_candidate_poses_kabsch();

  // Make the computed poses available to device.
  m_poseCandidates->UpdateDeviceFromHost();
}

void PreemptiveRansac_CUDA::prepare_inliers_for_optimisation()
{
  const Keypoint3DColour *keypointsData = m_keypointsImage->GetData(MEMORYDEVICE_CUDA);
  const ScorePrediction *predictionsData = m_predictionsImage->GetData(MEMORYDEVICE_CUDA);

  const size_t nbInliers = m_inliersIndicesBlock->dataSize;
  const int *inlierLinearisedIndicesData = m_inliersIndicesBlock->GetData(MEMORYDEVICE_CUDA);

  const size_t nbPoseCandidates = m_poseCandidates->dataSize;
  const PoseCandidate *poseCandidatesData = m_poseCandidates->GetData(MEMORYDEVICE_CUDA);

  // Grap pointers to the output storage.
  Vector4f *candidateCameraPoints = m_poseOptimisationCameraPoints->GetData(MEMORYDEVICE_CUDA);
  Keypoint3DColourCluster *candidateModes = m_poseOptimisationPredictedModes->GetData(MEMORYDEVICE_CUDA);

  dim3 blockSize(256);
  dim3 gridSize((nbInliers + blockSize.x - 1) / blockSize.x, nbPoseCandidates);

  ck_preemptive_ransac_prepare_inliers_for_optimisation<<<gridSize, blockSize>>>(keypointsData,
                                                                                 predictionsData,
                                                                                 inlierLinearisedIndicesData,
                                                                                 nbInliers,
                                                                                 poseCandidatesData,
                                                                                 nbPoseCandidates,
                                                                                 candidateCameraPoints,
                                                                                 candidateModes,
                                                                                 m_poseOptimisationInlierThreshold);
  ORcudaKernelCheck;

  // Compute the actual size of the buffers to avoid unnecessary copies.
  const uint32_t poseOptimisationBufferSize = nbInliers * nbPoseCandidates;
  m_poseOptimisationCameraPoints->dataSize = poseOptimisationBufferSize;
  m_poseOptimisationPredictedModes->dataSize = poseOptimisationBufferSize;

  // Make the inlier data available to the optimiser which is running on the CPU.
  m_poseOptimisationCameraPoints->UpdateHostFromDevice();
  m_poseOptimisationPredictedModes->UpdateHostFromDevice();
}

void PreemptiveRansac_CUDA::sample_inlier_candidates(bool useMask)
{
  const Vector2i imgSize = m_keypointsImage->noDims;
  const Keypoint3DColour *keypointsData = m_keypointsImage->GetData(MEMORYDEVICE_CUDA);
  const ScorePrediction *predictionsData = m_predictionsImage->GetData(MEMORYDEVICE_CUDA);

  int *inlierMaskData = m_inliersMaskImage->GetData(MEMORYDEVICE_CUDA);
  int *inlierIndicesData = m_inliersIndicesBlock->GetData(MEMORYDEVICE_CUDA);
  int *nbInlier_device = m_nbSampledInliers_device->GetData(MEMORYDEVICE_CUDA);
  CUDARNG *randomGenerators = m_randomGenerators->GetData(MEMORYDEVICE_CUDA);

  // Only if the number of inliers (host side) is zero, we reset the device number.
  // The assumption is that the number on device memory will remain in sync with the host
  // since only this method is allowed to modify it.
  if(m_inliersIndicesBlock->dataSize == 0)
  {
    ORcudaSafeCall(hipMemsetAsync(nbInlier_device, 0, sizeof(int)));
  }

  dim3 blockSize(128);
  dim3 gridSize((m_ransacInliersPerIteration + blockSize.x - 1) / blockSize.x);

  if(useMask)
  {
    ck_preemptive_ransac_sample_inliers<true><<<gridSize, blockSize>>>(keypointsData,
                                                                       predictionsData,
                                                                       imgSize,
                                                                       randomGenerators,
                                                                       inlierIndicesData,
                                                                       nbInlier_device,
                                                                       m_ransacInliersPerIteration,
                                                                       inlierMaskData);
    ORcudaKernelCheck;
  }
  else
  {
    ck_preemptive_ransac_sample_inliers<false><<<gridSize, blockSize>>>(keypointsData,
                                                                        predictionsData,
                                                                        imgSize,
                                                                        randomGenerators,
                                                                        inlierIndicesData,
                                                                        nbInlier_device,
                                                                        m_ransacInliersPerIteration);
    ORcudaKernelCheck;
  }

  // Update the number of inliers
  m_inliersIndicesBlock->dataSize = static_cast<size_t>(m_nbSampledInliers_device->GetElement(0, MEMORYDEVICE_CUDA));
}

void PreemptiveRansac_CUDA::update_candidate_poses()
{
  // The pose update is currently implemented by the base class, need to copy the relevant data to host memory.
  m_poseCandidates->UpdateHostFromDevice();

  PreemptiveRansac::update_candidate_poses();

  // The copy the updated poses back to the device.
  m_poseCandidates->UpdateDeviceFromHost();
}

//#################### PRIVATE MEMBER FUNCTIONS ####################

void PreemptiveRansac_CUDA::init_random()
{
  CUDARNG *randomGenerators = m_randomGenerators->GetData(MEMORYDEVICE_CUDA);

  // Initialize random states
  dim3 blockSize(256);
  dim3 gridSize((m_maxPoseCandidates + blockSize.x - 1) / blockSize.x);

  ck_reinit_rngs<<<gridSize, blockSize>>>(randomGenerators, m_maxPoseCandidates, m_rngSeed);
  ORcudaKernelCheck;
}

void PreemptiveRansac_CUDA::update_host_pose_candidates() const
{
  m_poseCandidates->UpdateHostFromDevice();
}

}
