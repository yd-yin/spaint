#include "hip/hip_runtime.h"
/**
 * grove: ScoreRelocaliser_CUDA.cu
 * Copyright (c) Torr Vision Group, University of Oxford, 2017. All rights reserved.
 */

#include "relocalisation/cuda/ScoreRelocaliser_CUDA.h"

#include <ITMLib/Engines/LowLevel/ITMLowLevelEngineFactory.h>
#include <ITMLib/Utils/ITMLibSettings.h>
using namespace ITMLib;

#include <itmx/base/MemoryBlockFactory.h>
using itmx::MemoryBlockFactory;

#include "clustering/ExampleClustererFactory.h"
#include "features/FeatureCalculatorFactory.h"
#include "forests/DecisionForestFactory.h"
#include "ransac/RansacFactory.h"
#include "relocalisation/shared/ScoreRelocaliser_Shared.h"
#include "reservoirs/ExampleReservoirsFactory.h"

using namespace tvgutil;

namespace grove {

//#################### CUDA KERNELS ####################

template <int TREE_COUNT>
__global__ void ck_score_relocaliser_get_predictions(const ScorePrediction *leafPredictions,
                                                     const ORUtils::VectorX<int, TREE_COUNT> *leafIndices,
                                                     ScorePrediction *outPredictions,
                                                     Vector2i imgSize,
                                                     int nbMaxPredictions)
{
  const int x = blockIdx.x * blockDim.x + threadIdx.x;
  const int y = blockIdx.y * blockDim.y + threadIdx.y;

  if (x >= imgSize.x || y >= imgSize.y) return;

  get_prediction_for_leaf_shared(leafPredictions, leafIndices, outPredictions, imgSize, nbMaxPredictions, x, y);
}

//#################### CONSTRUCTORS ####################

ScoreRelocaliser_CUDA::ScoreRelocaliser_CUDA(const SettingsContainer_CPtr& settings, const std::string& forestFilename)
  : ScoreRelocaliser(settings, forestFilename)
{
  // Instantiate the sub-algorithms knowing that we are running on the GPU.

  // Features.
  m_featureCalculator = FeatureCalculatorFactory::make_da_rgbd_patch_feature_calculator(ITMLibSettings::DEVICE_CUDA);

  // LowLevelEngine.
  m_lowLevelEngine.reset(ITMLowLevelEngineFactory::MakeLowLevelEngine(ITMLibSettings::DEVICE_CUDA));

  // Forest.
  m_scoreForest = DecisionForestFactory<DescriptorType, FOREST_TREE_COUNT>::make_forest(ITMLibSettings::DEVICE_CUDA,
                                                                                        m_forestFilename);

  // These variables have to be set here, since they depend on the forest that has just been loaded.
  m_reservoirsCount = m_scoreForest->get_nb_leaves();
  m_predictionsBlock = MemoryBlockFactory::instance().make_block<ScorePrediction>(m_reservoirsCount);

  // Reservoirs.
  m_exampleReservoirs = ExampleReservoirsFactory<ExampleType>::make_reservoirs(
      ITMLibSettings::DEVICE_CUDA, m_reservoirCapacity, m_reservoirsCount, m_rngSeed);

  // Clustering.
  m_exampleClusterer = ExampleClustererFactory<ExampleType, ClusterType, PredictionType::MAX_CLUSTERS>::make_clusterer(
      ITMLibSettings::DEVICE_CUDA, m_clustererSigma, m_clustererTau, m_maxClusterCount, m_minClusterSize);

  // P-RANSAC.
  m_preemptiveRansac = RansacFactory::make_preemptive_ransac(ITMLibSettings::DEVICE_CUDA, m_settings);

  // Clear internal state.
  reset();
}

//#################### PUBLIC VIRTUAL MEMBER FUNCTIONS ####################

ScorePrediction ScoreRelocaliser_CUDA::get_raw_prediction(uint32_t treeIdx, uint32_t leafIdx) const
{
  if (treeIdx >= m_scoreForest->get_nb_trees() || leafIdx >= m_scoreForest->get_nb_leaves_in_tree(treeIdx))
  {
    throw std::invalid_argument("Invalid tree or leaf index.");
  }

  return m_predictionsBlock->GetElement(leafIdx * m_scoreForest->get_nb_trees() + treeIdx, MEMORYDEVICE_CUDA);
}

//#################### PROTECTED VIRTUAL MEMBER FUNCTIONS ####################

void ScoreRelocaliser_CUDA::get_predictions_for_leaves(const LeafIndicesImage_CPtr &leafIndices,
                                                       const ScorePredictionsBlock_CPtr &leafPredictions,
                                                       ScorePredictionsImage_Ptr &outputPredictions) const
{
  const Vector2i imgSize = leafIndices->noDims;
  const LeafIndices *leafIndicesData = leafIndices->GetData(MEMORYDEVICE_CUDA);

  // Leaf predictions
  const ScorePrediction *leafPredictionsData = leafPredictions->GetData(MEMORYDEVICE_CUDA);

  // NOP after the first time.
  outputPredictions->ChangeDims(imgSize);
  ScorePrediction *outPredictionsData = outputPredictions->GetData(MEMORYDEVICE_CUDA);

  const dim3 blockSize(32, 32);
  const dim3 gridSize((imgSize.x + blockSize.x - 1) / blockSize.x, (imgSize.y + blockSize.y - 1) / blockSize.y);

  ck_score_relocaliser_get_predictions<<<gridSize, blockSize>>>(
      leafPredictionsData, leafIndicesData, outPredictionsData, imgSize, m_maxClusterCount);
  ORcudaKernelCheck;
}

} // namespace grove
