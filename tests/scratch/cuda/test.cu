#include "hip/hip_runtime.h"
#include "test.h"

#include "multiply.cuh"

__global__ void simple(float *x)
{
  int tid = threadIdx.x + blockDim.x * blockIdx.x;
  x[tid] = multiply(x[tid]);
}

void execute_simple_kernel(float *x, int numBlocks, int threadsPerBlock)
{
  simple<<<numBlocks,threadsPerBlock>>>(x);
}

__global__ void copy_1d_texture(hipTextureObject_t input, unsigned char *output)
{
  int tid = threadIdx.x + blockDim.x * blockIdx.x;
  output[tid] = tex1Dfetch<unsigned char>(input, tid);
}

void execute_copy_1d_texture_kernel(hipTextureObject_t input, unsigned char *output, int numBlocks, int threadsPerBlock)
{
  copy_1d_texture<<<numBlocks,threadsPerBlock>>>(input, output);
}

__global__ void copy_2d_texture(hipTextureObject_t input, unsigned char *output, int width, int height)
{
  int x = threadIdx.x + blockDim.x * blockIdx.x;
  int y = threadIdx.y + blockDim.y * blockIdx.y;
  if(x < width && y < height)
  {
    output[y * width + x] = tex2D<unsigned char>(input, x, y);
  }
}

void execute_copy_2d_texture_kernel(hipTextureObject_t input, unsigned char *output, int width, int height, int blockWidth, int blockHeight)
{
  dim3 dimBlock(blockWidth, blockHeight);
  dim3 dimGrid((width + dimBlock.x - 1) / dimBlock.x, (height + dimBlock.y - 1) / dimBlock.y);
  copy_2d_texture<<<dimGrid,dimBlock>>>(input, output, width, height);
}
